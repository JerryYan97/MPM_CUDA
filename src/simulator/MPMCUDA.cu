#include "hip/hip_runtime.h"
//
// Created by jiaruiyan on 5/28/21.
//
// Put Particle1 deformation gradient into python to check whether the stress1/2 has a problem.

#include "MPMSimulator.cuh"
#include <math.h>
#include <assert.h>
#include <thrust/device_vector.h>
#include "../../thirdparties/cudaSVD/svd3_cuda.h"

template<class T>
__device__ void Mat3x3Cofactor(const T* F, T* res){
    res[0] = F[4] * F[8] - F[5] * F[7];
    res[1] = F[5] * F[6] - F[3] * F[8];
    res[2] = F[3] * F[7] - F[4] * F[6];
    res[3] = F[2] * F[7] - F[1] * F[8];
    res[4] = F[0] * F[8] - F[2] * F[6];
    res[5] = F[1] * F[6] - F[0] * F[7];
    res[6] = F[1] * F[5] - F[2] * F[4];
    res[7] = F[2] * F[3] - F[0] * F[5];
    res[8] = F[0] * F[4] - F[1] * F[3];
}

template<class T>
__forceinline__
__device__ T Mat3x3Determinant(const T* X){
    return X[0] * (X[4] * X[8] - X[5] * X[7]) + X[1] * (X[3] * X[8] - X[5] * X[6]) + X[2] * (X[3] * X[7] - X[4] * X[6]);
}

template<class T>
__forceinline__
__device__ void MatMul3x3(const T* A, const T* B, T* C){
    C[0] = A[0] * B[0] + A[1] * B[3] + A[2] * B[6];
    C[1] = A[0] * B[1] + A[1] * B[4] + A[2] * B[7];
    C[2] = A[0] * B[2] + A[1] * B[5] + A[2] * B[8];
    C[3] = A[3] * B[0] + A[4] * B[3] + A[5] * B[6];
    C[4] = A[3] * B[1] + A[4] * B[4] + A[5] * B[7];
    C[5] = A[3] * B[2] + A[4] * B[5] + A[5] * B[8];
    C[6] = A[6] * B[0] + A[7] * B[3] + A[8] * B[6];
    C[7] = A[6] * B[1] + A[7] * B[4] + A[8] * B[7];
    C[8] = A[6] * B[2] + A[7] * B[5] + A[8] * B[8];
}

template<class T>
__forceinline__
__device__ void MatTranspose(const T* x, T* transpose) {
    transpose[0]=x[0]; transpose[1]=x[3]; transpose[2]=x[6];
    transpose[3]=x[1]; transpose[4]=x[4]; transpose[5]=x[7];
    transpose[6]=x[2]; transpose[7]=x[5]; transpose[8]=x[8];
}

__device__ void FixedCorotatedPStressSigma(float sigma1, float sigma2, float sigma3,
                                           float mu, float lambda,
                                           float& dig1, float& dig2, float& dig3){
    dig1 = 2.f * mu * (sigma1 - 1.f) + lambda * (sigma1 * sigma2 * sigma3 - 1.f) * sigma2 * sigma3;
    dig2 = 2.f * mu * (sigma2 - 1.f) + lambda * (sigma1 * sigma2 * sigma3 - 1.f) * sigma1 * sigma3;
    dig3 = 2.f * mu * (sigma3 - 1.f) + lambda * (sigma1 * sigma2 * sigma3 - 1.f) * sigma1 * sigma2;
}

__device__ void FixedCorotatedPStress(float F11, float F12, float F13,
                                      float F21, float F22, float F23,
                                      float F31, float F32, float F33,
                                      float mu, float lambda,
                                      float &P11, float &P12, float &P13,
                                      float &P21, float &P22, float &P23,
                                      float &P31, float &P32, float &P33){
    float U11, U12, U13, U21, U22, U23, U31, U32, U33;
    float V11, V12, V13, V21, V22, V23, V31, V32, V33;
    float S11, S22, S33;
    float dig1, dig2, dig3;

    /*
    if (abs(F11 - 0.f) < FLT_EPSILON){
        F11 = 0.f;
    }
    if (abs(F12 - 0.f) < FLT_EPSILON){
        F12 = 0.f;
    }
    if (abs(F13 - 0.f) < FLT_EPSILON){
        F13 = 0.f;
    }
    if (abs(F21 - 0.f) < FLT_EPSILON){
        F21 = 0.f;
    }
    if (abs(F22 - 0.f) < FLT_EPSILON){
        F22 = 0.f;
    }
    if (abs(F23 - 0.f) < FLT_EPSILON){
        F23 = 0.f;
    }
    if (abs(F31 - 0.f) < FLT_EPSILON){
        F31 = 0.f;
    }
    if (abs(F32 - 0.f) < FLT_EPSILON){
        F32 = 0.f;
    }
    if (abs(F33 - 0.f) < FLT_EPSILON){
        F33 = 0.f;
    }
    */

    svd(F11, F12, F13, F21, F22, F23, F31, F32, F33,
        U11, U12, U13, U21, U22, U23, U31, U32, U33,
        S11, S22, S33,
        V11, V12, V13, V21, V22, V23, V31, V32, V33);
    FixedCorotatedPStressSigma(S11, S22, S33, mu, lambda, dig1, dig2, dig3);

    float V[9] = {V11, V12, V13,
                  V21, V22, V23,
                  V31, V32, V33};
    float U[9] = {U11, U12, U13,
                  U21, U22, U23,
                  U31, U32, U33};

    /*
    assert(S11 > 0.f || abs(S11) <= 1e-6);
    assert(S22 > 0.f || abs(S22) <= 1e-6);
    assert(S11 > S22 || abs(S11 - S22) <= 1e-6);
    assert(S22 > abs(S33) || abs(S22 - abs(S33)) < 1e-6);
    */

    assert(Mat3x3Determinant(U) >= 0.f);
    assert(Mat3x3Determinant(V) >= 0.f);



    float P_sigma[9] = {dig1, 0.f, 0.f,
                        0.f, dig2, 0.f,
                        0.f, 0.f, dig3};
    float V_transpose[9];
    float tmpMat[9] = {0.f};
    float res[9] = {0.f};
    MatTranspose(V, V_transpose);
    MatMul3x3(P_sigma, V_transpose, tmpMat);
    MatMul3x3(U, tmpMat, res);
    P11 = res[0];
    P12 = res[1];
    P13 = res[2];
    P21 = res[3];
    P22 = res[4];
    P23 = res[5];
    P31 = res[6];
    P32 = res[7];
    P33 = res[8];
}

template<class T>
__device__ void MatAdd(const T* m1, const T* m2, T* mAdd, int eleNum){
    for (int i = 0; i < eleNum; ++i){
        mAdd[i] = m1[i] + m2[i];
    }
}

template<class T>
__forceinline__
__device__ __host__ void MatVelMul3x3x3x1(const T* X, const T* V, T* R)
{
    R[0] = X[0] * V[0] + X[1] * V[1] + X[2] * V[2];
    R[1] = X[3] * V[0] + X[4] * V[1] + X[5] * V[2];
    R[2] = X[6] * V[0] + X[7] * V[1] + X[8] * V[2];
}


template<class T>
__device__ void ScalarMatMul(const T scalar, const T* mat, T* res, int matEleNum){
    for (int i = 0; i < matEleNum; ++i){
        res[i] = scalar * mat[i];
    }
}

template<class T>
__forceinline__
__device__ void OuterProduct(const T* v1, const T* v2, T* res){
    res[0] = v1[0] * v2[0];
    res[1] = v1[0] * v2[1];
    res[2] = v1[0] * v2[2];
    res[3] = v1[1] * v2[0];
    res[4] = v1[1] * v2[1];
    res[5] = v1[1] * v2[2];
    res[6] = v1[2] * v2[0];
    res[7] = v1[2] * v2[1];
    res[8] = v1[2] * v2[2];
}

template<class T>
__device__ void FixedCorotatedStress2(const T* F, const T mu, const T lambda, T* P){
    T F_invT[9] = {0.0};
    Mat3x3Cofactor(F, F_invT);
    T J = Mat3x3Determinant(F);

    float U11, U12, U13, U21, U22, U23, U31, U32, U33;
    float V11, V12, V13, V21, V22, V23, V31, V32, V33;
    float S11, S22, S33;

    svd(F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
        U11, U12, U13, U21, U22, U23, U31, U32, U33,
        S11, S22, S33,
        V11, V12, V13, V21, V22, V23, V31, V32, V33);

    float V[9] = {V11, V12, V13,
                  V21, V22, V23,
                  V31, V32, V33};
    float U[9] = {U11, U12, U13,
                  U21, U22, U23,
                  U31, U32, U33};
    float R[9] = {0.f};
    float term1[9] = {0.f};
    float term2[9] = {0.f};

    float V_transpose[9];
    MatTranspose(V, V_transpose);
    MatMul3x3(U, V_transpose, R);

    float min_R[9] = {0.f};
    ScalarMatMul(-1.f, R, min_R, 9);

    float F_min_R[9] = {0.f};
    MatAdd(F, min_R, F_min_R, 9);
    ScalarMatMul(2.f * mu, F_min_R, term1, 9);

    float JF_invT[9] = {0.f};
    Mat3x3Cofactor(F, JF_invT);
    ScalarMatMul(lambda * (J - 1.f), JF_invT, term2, 9);

    MatAdd(term1, term2, P, 9);
}

__device__ double BSplineInterpolation1DDerivative(const double x){
    if (x > -0.5 && x < 0.5){
        return -2.0 * x;
    }else if (x >= 0.5 && x < 1.5){
        return x - 1.5;
    }else if (x > -1.5 && x <= -0.5){
        return 1.5 + x;
    }else{
        return 0.0;
    }
}

__device__ double BSplineInterpolation1D(const double x){
    double abs_x = abs(x);
    if (abs_x >= 0 && abs_x < 0.5){
        return 0.75 - abs_x * abs_x;
    }
    else if (abs_x >= 0.5 && abs_x < 1.5){
        return 0.5 * (1.5 - abs_x) * (1.5 - abs_x);
    }
    else{
        return 0.0;
    }
}

__device__ void BSplineInterpolationGradient(const double xp[3], const double xi[3], const double h,
                                             double& gx, double& gy, double&gz){
    double h_inv = 1.0 / h;
    double i1 = h_inv * (xp[0] - xi[0]);
    double i2 = h_inv * (xp[1] - xi[1]);
    double i3 = h_inv * (xp[2] - xi[2]);
    gx = h_inv * BSplineInterpolation1DDerivative(i1) * BSplineInterpolation1D(i2) * BSplineInterpolation1D(i3);
    gy = h_inv * BSplineInterpolation1D(i1) * BSplineInterpolation1DDerivative(i2) * BSplineInterpolation1D(i3);
    gz = h_inv * BSplineInterpolation1D(i1) * BSplineInterpolation1D(i2) * BSplineInterpolation1DDerivative(i3);
}

__device__ double BSplineInterpolation(const double xp[3], const double xi[3], const double h){
    // printf("Interpolation:(%f, %f, %f)\n", (xp[0] - xi[0]) / h, (xp[1] - xi[1]) / h, (xp[2] - xi[2]) / h);
    return BSplineInterpolation1D((xp[0] - xi[0]) / h) *
           BSplineInterpolation1D((xp[1] - xi[1]) / h) *
           BSplineInterpolation1D((xp[2] - xi[2]) / h);
}

__global__ void P2G(unsigned int pNum,
                    double* pPosVec, double* pMassVec, double* pVelVec, double* pDGVec,
                    double* pVolVec, double* pForceVec, // int* pAttentionLabel,
                    double gOriCorner_x, double gOriCorner_y, double gOriCorner_z, int* gAttentionIdx,
                    unsigned int gNodeNumDim, double h, double mu, double lambda,
                    double* gNodeMassVec, double* gNodeTmpMotVec, double* gElasticityForceVec){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < pNum){
        double pos[3] = {pPosVec[i * 3], pPosVec[i * 3 + 1], pPosVec[i * 3 + 2]};
        double m = pMassVec[i];
        double vel[3] = {pVelVec[i * 3], pVelVec[i * 3 + 1], pVelVec[i * 3 + 2]};
        float tmpDeformationGradient[9] = {float(pDGVec[9 * i]), float(pDGVec[9 * i + 1]), float(pDGVec[9 * i + 2]),
                                           float(pDGVec[9 * i + 3]), float(pDGVec[9 * i + 4]), float(pDGVec[9 * i + 5]),
                                           float(pDGVec[9 * i + 6]), float(pDGVec[9 * i + 7]), float(pDGVec[9 * i + 8])};
        float stress[9] = {0.f};
        FixedCorotatedPStress(tmpDeformationGradient[0], tmpDeformationGradient[1], tmpDeformationGradient[2],
                              tmpDeformationGradient[3], tmpDeformationGradient[4], tmpDeformationGradient[5],
                              tmpDeformationGradient[6], tmpDeformationGradient[7], tmpDeformationGradient[8],
                              float(mu), float(lambda),
                              stress[0], stress[1], stress[2],
                              stress[3], stress[4], stress[5],
                              stress[6], stress[7], stress[8]);
        float stress2[9] = {0.f};
        FixedCorotatedStress2(tmpDeformationGradient, float(mu), float(lambda), stress2);

        // float tar_stress[9] = {0.0};
        // Times_Rotated_dP_dF_FixedCorotated(float(mu), float(lambda), tmpDeformationGradient, );

        float F_transpose[9] = {0.f};
        MatTranspose(tmpDeformationGradient, F_transpose);
        float tmpMat[9] = {0.f};
        // MatMul3x3(stress, F_transpose, tmpMat);
        MatMul3x3(stress2, F_transpose, tmpMat);

        float mVol(pVolVec[i]);
        ScalarMatMul(mVol, tmpMat, tmpMat, 9);
        /*
        if (i == 1){
            printf("P2G: Tmp Mat of particle 1:\n");
            printf("[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n",
                   tmpMat[0], tmpMat[1], tmpMat[2],
                   tmpMat[3], tmpMat[4], tmpMat[5],
                   tmpMat[6], tmpMat[7], tmpMat[8]);
            printf("P2G: Stress of particle 1:\n");
            printf("[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n",
                   stress[0], stress[1], stress[2],
                   stress[3], stress[4], stress[5],
                   stress[6], stress[7], stress[8]);
            printf("P2G: Stress2 of particle 1:\n");
            printf("[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n",
                   stress2[0], stress2[1], stress2[2],
                   stress2[3], stress2[4], stress2[5],
                   stress2[6], stress2[7], stress2[8]);
            printf("P2G: F1:\n");
            printf("[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n",
                   tmpDeformationGradient[0], tmpDeformationGradient[1], tmpDeformationGradient[2],
                   tmpDeformationGradient[3], tmpDeformationGradient[4], tmpDeformationGradient[5],
                   tmpDeformationGradient[6], tmpDeformationGradient[7], tmpDeformationGradient[8]);
            printf("P2G: F1 determinant:%f\n", Mat3x3Determinant(tmpDeformationGradient));
            printf("mu:%f, lambda:%f\n", mu, lambda);
        }
        */

        int b_idx_x = max(0, int((pos[0] - gOriCorner_x - 0.5 * h) / h));
        int b_idx_y = max(0, int((pos[1] - gOriCorner_y - 0.5 * h) / h));
        int b_idx_z = max(0, int((pos[2] - gOriCorner_z - 0.5 * h) / h));
        double t_w = 0.0;
        double t_m = 0.0;
        float t_f_x = 0.0;
        float t_f_y = 0.0;
        float t_f_z = 0.0;
        for (int idx_x_offset = 0; idx_x_offset < 3; ++idx_x_offset){
            for (int idx_y_offset = 0; idx_y_offset < 3; ++idx_y_offset){
                for (int idx_z_offset = 0; idx_z_offset < 3; ++idx_z_offset){
                    int idx_x = b_idx_x + idx_x_offset;
                    int idx_y = b_idx_y + idx_y_offset;
                    int idx_z = b_idx_z + idx_z_offset;
                    double b_pos[3] = {gOriCorner_x + idx_x * h,
                                       gOriCorner_y + idx_y * h,
                                       gOriCorner_z + idx_z * h};
                    // printf("b_pos:(%f, %f, %f)\n", b_pos[0], b_pos[1], b_pos[2]);
                    double w = BSplineInterpolation(pos, b_pos, h);
                    int g_idx = idx_z * gNodeNumDim * gNodeNumDim + idx_y * gNodeNumDim + idx_x;
                    assert(g_idx < gNodeNumDim * gNodeNumDim * gNodeNumDim);
                    assert(g_idx >= 0);

                    atomicAdd(&gNodeMassVec[g_idx], w * m);
                    t_m += w * m;

                    atomicAdd(&gNodeTmpMotVec[3 * g_idx], w * m * vel[0]);
                    atomicAdd(&gNodeTmpMotVec[3 * g_idx + 1], w * m * vel[1]);
                    atomicAdd(&gNodeTmpMotVec[3 * g_idx + 2], w * m * vel[2]);
                    t_w += w;

                    if (i == 1){
                        // printf("P109743 related node Mot(g_idx=%d)=[%f, %f, %f] w=%f.\n", g_idx,
                        //        w * m * vel[0], w * m * vel[1], w * m * vel[2], w);
                        gAttentionIdx[idx_x_offset * 9 + idx_y_offset * 3 + idx_z_offset] = g_idx;
                    }

                    // Transfer elasticity force to grid.
                    double grad_wip[3] = {0.0};
                    float tmpForce[3] = {0.f};
                    BSplineInterpolationGradient(pos, b_pos, h, grad_wip[0], grad_wip[1], grad_wip[2]);
                    float grad_wip_f[3] = {static_cast<float>(grad_wip[0]), static_cast<float>(grad_wip[1]), static_cast<float>(grad_wip[2])};
                    MatVelMul3x3x3x1(tmpMat, grad_wip_f, tmpForce);
                    atomicAdd(&gElasticityForceVec[3 * g_idx], -tmpForce[0]);
                    atomicAdd(&gElasticityForceVec[3 * g_idx + 1], -tmpForce[1]);
                    atomicAdd(&gElasticityForceVec[3 * g_idx + 2], -tmpForce[2]);
                    /*
                    if (i == 1){
                        printf("Particle %d contributes force to g_idx = %d:[%f, %f, %f]\n", i, g_idx, -tmpForce[0], -tmpForce[1], -tmpForce[2]);
                        printf("Particle %d contributes grad_wip = %d:[%f, %f, %f]\n", i, g_idx, grad_wip_f[0], grad_wip_f[1], grad_wip_f[2]);
                        // pAttentionLabel[i] = 1;
                    }
                    */

                    t_f_x -= tmpForce[0];
                    t_f_y -= tmpForce[1];
                    t_f_z -= tmpForce[2];

                    /*
                    if (i == 0){
                        printf("grad_wip_f:(%f, %f, %f)\n", grad_wip_f[0], grad_wip_f[1], grad_wip_f[2]);
                        printf("tmpForce:(%f, %f, %f)\n", tmpForce[0], tmpForce[1], tmpForce[2]);
                        printf("total force:(%f, %f, %f)\n", t_f_x, t_f_y, t_f_z);
                    }
                    */
                }
            }
        }
        pForceVec[3 * i] = double(t_f_x);
        pForceVec[3 * i + 1] = double(t_f_y);
        pForceVec[3 * i + 2] = double(t_f_z);
        /*
        if (abs(t_f_x) > 1.0 || abs(t_f_y) > 1.0 || abs(t_f_z) > 1.0){
            printf("Particle %d contributes force:[%f, %f, %f]\n", i, t_f_x, t_f_y, t_f_z);
            // pAttentionLabel[i] = 1;
        }

        if (i == 1){
            printf("P1 Elasticity Force Contribution:(%f, %f, %f)\n", t_f_x, t_f_y, t_f_z);
        }
        */

        assert(abs(t_w - 1.0) < 0.001);
        // assert(abs(t_m - 1.0) < 0.001);
    }
}

__global__ void VelUpdate(unsigned int gNum, double dt, double ext_gravity,
                          double lower_x, double lower_y, double lower_z,
                          double upper_x, double upper_y, double upper_z,
                          double gOriCorner_x, double gOriCorner_y, double gOriCorner_z,
                          unsigned int gNodeNumDim, double h,
                          double* gMassVec, double* gVelMotVec, double* gForceVec){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < gNum){
        double mass = gMassVec[i];
        if (abs(mass) < DBL_EPSILON){
            gMassVec[i] = 0.0;
            gVelMotVec[3 * i] = 0.0;
            gVelMotVec[3 * i + 1] = 0.0;
            gVelMotVec[3 * i + 2] = 0.0;
            gForceVec[3 * i] = 0.0;
            gForceVec[3 * i + 1] = 0.0;
            gForceVec[3 * i + 2] = 0.0;
        }
        else{
            // Calculate velocity from momentum.
            gVelMotVec[3 * i] = gVelMotVec[3 * i] / mass;
            gVelMotVec[3 * i + 1] = gVelMotVec[3 * i + 1] / mass;
            gVelMotVec[3 * i + 2] = gVelMotVec[3 * i + 2] / mass;

            // Include gravity into velocity.
            gVelMotVec[3 * i + 1] = gVelMotVec[3 * i + 1] + ext_gravity * dt;

            // Include elasticity force into velocity.
            gVelMotVec[3 * i] += (dt * gForceVec[3 * i] / mass);
            gVelMotVec[3 * i + 1] += (dt * gForceVec[3 * i + 1] / mass);
            gVelMotVec[3 * i + 2] += (dt * gForceVec[3 * i + 2] / mass);

            // Deal with Boundary condition.
            int idx_x = i % int(gNodeNumDim);
            int idx_y = ((i - idx_x) / int(gNodeNumDim)) % int(gNodeNumDim);
            int idx_z = ((i - idx_x) / int(gNodeNumDim) - idx_y) / int(gNodeNumDim);
            double grid_node_pos[3] = {gOriCorner_x + idx_x * h,
                                       gOriCorner_y + idx_y * h,
                                       gOriCorner_z + idx_z * h};
            if (grid_node_pos[0] <= lower_x || grid_node_pos[0] >= upper_x ||
                grid_node_pos[1] <= lower_y || grid_node_pos[1] >= upper_y ||
                grid_node_pos[2] <= lower_z || grid_node_pos[2] >= upper_z){
                gVelMotVec[3 * i] = 0.0;
                gVelMotVec[3 * i + 1] = 0.0;
                gVelMotVec[3 * i + 2] = 0.0;
            }
        }
    }
}

__global__ void InterpolateAndMove(unsigned int pNum, double dt,
                                   double* pPosVec, double* pVelVec, double* pDGVec,
                                   double gOriCorner_x, double gOriCorner_y, double gOriCorner_z,
                                   unsigned int gNodeNumDim, double h, double* gNodeVelVec){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < pNum){
        double pos[3] = {pPosVec[i * 3], pPosVec[i * 3 + 1], pPosVec[i * 3 + 2]};
        int b_idx_x = max(0, int((pos[0] - gOriCorner_x - 0.5 * h) / h));
        int b_idx_y = max(0, int((pos[1] - gOriCorner_y - 0.5 * h) / h));
        int b_idx_z = max(0, int((pos[2] - gOriCorner_z - 0.5 * h) / h));
        double t_w = 0.0;
        double t_vel_x = 0.0;
        double t_vel_y = 0.0;
        double t_vel_z = 0.0;
        for (int idx_x_offset = 0; idx_x_offset < 3; ++idx_x_offset){
            for (int idx_y_offset = 0; idx_y_offset < 3; ++idx_y_offset){
                for (int idx_z_offset = 0; idx_z_offset < 3; ++idx_z_offset){
                    int idx_x = b_idx_x + idx_x_offset;
                    int idx_y = b_idx_y + idx_y_offset;
                    int idx_z = b_idx_z + idx_z_offset;
                    double b_pos[3] = {gOriCorner_x + idx_x * h,
                                       gOriCorner_y + idx_y * h,
                                       gOriCorner_z + idx_z * h};

                    double w = BSplineInterpolation(pos, b_pos, h);
                    int g_idx = idx_z * gNodeNumDim * gNodeNumDim + idx_y * gNodeNumDim + idx_x;
                    assert(g_idx < gNodeNumDim * gNodeNumDim * gNodeNumDim);
                    assert(g_idx >= 0);

                    pVelVec[3 * i] += w * gNodeVelVec[3 * g_idx];
                    pVelVec[3 * i + 1] += w * gNodeVelVec[3 * g_idx + 1];
                    pVelVec[3 * i + 2] += w * gNodeVelVec[3 * g_idx + 2];

                    t_vel_x += gNodeVelVec[3 * g_idx];
                    t_vel_y += gNodeVelVec[3 * g_idx + 1];
                    t_vel_z += gNodeVelVec[3 * g_idx + 2];
                    // printf("gNode vel:[%f, %f, %f]\n", gNodeVelVec[3 * g_idx], gNodeVelVec[3 * g_idx + 1], gNodeVelVec[3 * g_idx + 2]);
                    t_w += w;
                }
            }
        }
        assert(abs(t_w - 1.0) < 0.0001);
        double vel_p[3] = {pVelVec[3 * i], pVelVec[3 * i + 1], pVelVec[3 * i + 2]};

        // Update deformation gradient
        double grad_v[9] = {0.0};
        for (int idx_x_offset = 0; idx_x_offset < 3; ++idx_x_offset){
            for (int idx_y_offset = 0; idx_y_offset < 3; ++idx_y_offset){
                for (int idx_z_offset = 0; idx_z_offset < 3; ++idx_z_offset){
                    int idx_x = b_idx_x + idx_x_offset;
                    int idx_y = b_idx_y + idx_y_offset;
                    int idx_z = b_idx_z + idx_z_offset;
                    double b_pos[3] = {gOriCorner_x + idx_x * h,
                                       gOriCorner_y + idx_y * h,
                                       gOriCorner_z + idx_z * h};
                    int g_idx = idx_z * gNodeNumDim * gNodeNumDim + idx_y * gNodeNumDim + idx_x;
                    assert(g_idx < gNodeNumDim * gNodeNumDim * gNodeNumDim);
                    assert(g_idx >= 0);

                    double grad_wip[3] = {0.0};
                    BSplineInterpolationGradient(pos, b_pos, h, grad_wip[0], grad_wip[1], grad_wip[2]);

                    double vi[3] = {gNodeVelVec[3 * g_idx], gNodeVelVec[3 * g_idx + 1], gNodeVelVec[3 * g_idx + 2]};

                    /*
                    if (i == 109743){
                        printf("P109743 related node vel(g_idx=%d)=[%f, %f, %f]:\n", g_idx, vi[0], vi[1], vi[2]);
                    }
                    */

                    double add_mat[9] = {0.0};
                    OuterProduct(vi, grad_wip, add_mat);

                    double tmp_grad_v[9];
                    memcpy(tmp_grad_v, grad_v, sizeof(double) * 9);
                    MatAdd(tmp_grad_v, add_mat, grad_v, 9);
                }
            }
        }
        double Fp[9] = {pDGVec[9 * i], pDGVec[9 * i + 1], pDGVec[9 * i + 2],
                        pDGVec[9 * i + 3], pDGVec[9 * i + 4], pDGVec[9 * i + 5],
                        pDGVec[9 * i + 6], pDGVec[9 * i + 7], pDGVec[9 * i + 8]};
        double leftMat[9] = {1.0, 0.0, 0.0,
                              0.0, 1.0, 0.0,
                              0.0, 0.0, 1.0};
        double tmp_leftMat[9] = {1.0, 0.0, 0.0,
                             0.0, 1.0, 0.0,
                             0.0, 0.0, 1.0};
        ScalarMatMul(dt, grad_v, grad_v, 9);
        MatAdd(tmp_leftMat, grad_v, leftMat, 9);
        double tmp_Fp[9];
        memcpy(tmp_Fp, Fp, sizeof(double) * 9);
        MatMul3x3(leftMat, tmp_Fp, Fp);
        pDGVec[9 * i] = Fp[0];
        pDGVec[9 * i + 1] = Fp[1];
        pDGVec[9 * i + 2] = Fp[2];
        pDGVec[9 * i + 3] = Fp[3];
        pDGVec[9 * i + 4] = Fp[4];
        pDGVec[9 * i + 5] = Fp[5];
        pDGVec[9 * i + 6] = Fp[6];
        pDGVec[9 * i + 7] = Fp[7];
        pDGVec[9 * i + 8] = Fp[8];

        /*
        if (i == 5252144){
            printf("Updated: F5252144:\n");
            printf("[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n",
                   Fp[0], Fp[1], Fp[2],
                   Fp[3], Fp[4], Fp[5],
                   Fp[6], Fp[7], Fp[8]);
            printf("Updated: F5252144 determinant:%f\n", Mat3x3Determinant(Fp));
        }
        */

        // Apply velocity
        pPosVec[3 * i] += dt * vel_p[0];
        pPosVec[3 * i + 1] += dt * vel_p[1];
        pPosVec[3 * i + 2] += dt * vel_p[2];
        // printf("total vel:[%f, %f, %f]\n", t_vel_x, t_vel_y, t_vel_z);
        // printf("pVel:[%f, %f, %f]\n", t_vel_x, t_vel_y, t_vel_z);
    }
}

__global__ void FindAllRelatedParticles(unsigned int pNum, double* pPosVec,
                                        double gOriCorner_x, double gOriCorner_y, double gOriCorner_z,
                                        double h, unsigned int gNodeNumDim,
                                        int* gAttentionIdx,
                                        int* pAttentionParticleIdx){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < pNum){
        double pos[3] = {pPosVec[i * 3], pPosVec[i * 3 + 1], pPosVec[i * 3 + 2]};
        int b_idx_x = max(0, int((pos[0] - gOriCorner_x - 0.5 * h) / h));
        int b_idx_y = max(0, int((pos[1] - gOriCorner_y - 0.5 * h) / h));
        int b_idx_z = max(0, int((pos[2] - gOriCorner_z - 0.5 * h) / h));

        for (int idx_x_offset = 0; idx_x_offset < 3; ++idx_x_offset){
            for (int idx_y_offset = 0; idx_y_offset < 3; ++idx_y_offset){
                for (int idx_z_offset = 0; idx_z_offset < 3; ++idx_z_offset){
                    int idx_x = b_idx_x + idx_x_offset;
                    int idx_y = b_idx_y + idx_y_offset;
                    int idx_z = b_idx_z + idx_z_offset;
                    int g_idx = idx_z * gNodeNumDim * gNodeNumDim + idx_y * gNodeNumDim + idx_x;
                    for (int j = 0; j < 27; ++j){
                        if (gAttentionIdx[j] == g_idx){
                            pAttentionParticleIdx[i] = 1.0;
                            return;
                        }
                    }
                }
            }
        }
    }
}

__global__ void FindOutBoundParticles(unsigned int pNum, double* pPosVec,
                                      double minBoundX, double minBoundY, double minBoundZ,
                                      double maxBoundX, double maxBoundY, double maxBoundZ,
                                      int* pAttentionParticleLabel){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < pNum){
        double pos[3] = {pPosVec[i * 3], pPosVec[i * 3 + 1], pPosVec[i * 3 + 2]};
        if (pos[0] < minBoundX || pos[0] > maxBoundX ||
            pos[1] < minBoundY || pos[1] > maxBoundY ||
            pos[2] < minBoundZ || pos[2] > maxBoundZ){
            pAttentionParticleLabel[i] = 1;
        }
        else{
            pAttentionParticleLabel[i] = 0;
        }
    }
}

/* This can be done freely during P2G:
void FindRelatedGrid(int tarIdx, double gOriCorner_x, double gOriCorner_y, double gOriCorner_z,
                     double h, std::vector<int>& gAttentionIdx){

}
*/

void MPMSimulator::step() {

    // 0. Check each particles is affected by 9 grid nodes and within the grid.
    hipError_t err = hipSuccess;

    // 1. Clean grid data.
    err = hipMemset(mGrid.nodeMassVec, 0, mGrid.massVecByteSize);
    if(err != hipSuccess){
        std::cerr << "Clean grid mass error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    err = hipMemset(mGrid.nodeVelVec, 0, mGrid.velVecByteSize);
    if (err != hipSuccess){
        std::cerr << "Clean grid velocity error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    err = hipMemset(mGrid.nodeForceVec, 0, mGrid.forceVecByteSize);
    if (err != hipSuccess){
        std::cerr << "Clean grid force error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }

#ifdef DEBUG
    std::cout << "********* Frame " << current_frame << " starts **********" << std::endl << std::endl;
    double standard_speed = ext_gravity * t;
    // Check particle 1 pos and vel:
    std::cout << "n vel:[" << mParticles.particleVelVec[1 * 3] << " " << mParticles.particleVelVec[1 * 3 + 1] << " " << mParticles.particleVelVec[1 * 3 + 2] << "]" << std::endl;
    std::cout << "n pos:[" << mParticles.particlePosVec[1 * 3] << " " << mParticles.particlePosVec[1 * 3 + 1] << " " << mParticles.particlePosVec[1 * 3 + 2] << "]" << std::endl << std::endl;

#endif
    double* pForceVec;
    err = hipMalloc(&pForceVec, mParticles.particleNum * sizeof(double) * 3);
    if (err != hipSuccess){
        std::cerr << "Allocate particle force vector error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    int* gAttentionIdx;
    err = hipMalloc(&gAttentionIdx, 27 * sizeof(int));
    if (err != hipSuccess){
        std::cerr << "Allocate attention idx error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    // 2. Transfer mass to the grid.
    // 3. Transfer velocity(Momentum) to the grid.
    int pThreadsPerBlock = 256;
    int pBlocksPerGrid = (mParticles.particleNum + pThreadsPerBlock - 1) / pThreadsPerBlock;
    P2G<<<pBlocksPerGrid, pThreadsPerBlock>>>(mParticles.particleNum,
                                            mParticles.pPosVecGRAM,
                                            mParticles.pMassVecGRAM,
                                            mParticles.pVelVecGRAM,
                                            mParticles.pDeformationGradientGRAM,
                                            mParticles.pVolVecGRAM,
                                            pForceVec,
                                            mGrid.originCorner[0], mGrid.originCorner[1], mGrid.originCorner[2],
                                            gAttentionIdx,
                                            mGrid.nodeNumDim,
                                            mGrid.h, mParticles.mMaterialVec[0].mMu, mParticles.mMaterialVec[0].mLambda,
                                            mGrid.nodeMassVec,
                                            mGrid.nodeVelVec,
                                            mGrid.nodeForceVec);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch P2G kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

#ifdef DEBUG
    double pos0_y_init = mParticles.particlePosVec[1];
    double vel0_y_init = mParticles.particleVelVec[1];
    double vel_energy_init = 0.5 * 1.0 * vel0_y_init * vel0_y_init;

    // Check whether grid mass is equal to particles mass.
    double* h_gMassVec = (double*)malloc(mGrid.massVecByteSize);
    err = hipMemcpy(h_gMassVec, mGrid.nodeMassVec, mGrid.massVecByteSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy grid mass memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    thrust::device_vector<double> gMass(h_gMassVec, h_gMassVec + mGrid.massVecByteSize / sizeof(double));
    thrust::device_vector<double> pMass(mParticles.particleMassVec.begin(), mParticles.particleMassVec.end());

    double gSum = thrust::reduce(gMass.begin(), gMass.end());
    double pSum = thrust::reduce(pMass.begin(), pMass.end());
    if (abs(gSum - pSum) > 0.0001){
        std::cerr << "Mass is different between Grid and particles after P2G." << std::endl;
        std::cerr << "gSum:" << gSum << " pSum:" << pSum << std::endl;
        exit(1);
    }
    free(h_gMassVec);

    // Check force on the grid.
    double* h_gForceVec = (double*)malloc(mGrid.forceVecByteSize);
    err = hipMemcpy(h_gForceVec, mGrid.nodeForceVec, mGrid.forceVecByteSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy grid force memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    int* h_gAttentionIdx = (int*)malloc(sizeof(int) * 27);
    err = hipMemcpy(h_gAttentionIdx, gAttentionIdx, sizeof(int) * 27, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy attentionIdx memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    for (int i = 0; i < 27; ++i){
        int g_idx = h_gAttentionIdx[i];
        std::cout << "f(g_idx=" << g_idx << ")" << "=[" << h_gForceVec[g_idx * 3] << ", " << h_gForceVec[g_idx * 3 + 1] << ", " << h_gForceVec[g_idx * 3 + 2] << "]" << std::endl;
    }

    int* d_pAttentionLabel;
    err = hipMalloc(&d_pAttentionLabel, mParticles.particleNum * sizeof(int));
    if (err != hipSuccess){
        std::cerr << "Allocate pAttentionLabel memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    err = hipMemset(d_pAttentionLabel, 0, mParticles.particleNum * sizeof(int));
    if (err != hipSuccess){
        std::cerr << "Clean AttentionLabel error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    /*
    FindAllRelatedParticles<<<pBlocksPerGrid, pThreadsPerBlock>>>(mParticles.particleNum,
                                                                  mParticles.pPosVecGRAM,
                                                                  mGrid.originCorner[0],
                                                                  mGrid.originCorner[1],
                                                                  mGrid.originCorner[2],
                                                                  mGrid.h,
                                                                  mGrid.nodeNumDim,
                                                                  gAttentionIdx,
                                                                  pAttentionLabel);
    */

    double* h_pForce = (double*)malloc(sizeof(double) * mParticles.particleNum * 3);
    err = hipMemcpy(h_pForce, pForceVec, sizeof(double) * mParticles.particleNum * 3, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy pForceVec memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    /* Attention Nodes:
    for (int i = 0; i < mParticles.particleNum; ++i){
        if (h_pAttentionLabel[i] == 1){
            std::cout << "Related nodes are affected by particle_idx = " << i << std::endl;
            std::cout << "f(p_idx = " << i << ") = " << "[" << h_pForce[i * 3] << ", " << h_pForce[i * 3 + 1] << ", " << h_pForce[i * 3 + 2] << "]" << std::endl;
            h_pAttentionIdx.push_back(i);
        }
    }
    */
#endif

    // 4. Calculate the velocity.
    // 5. Apply gravity.
    int gNum = mGrid.nodeNumDim * mGrid.nodeNumDim * mGrid.nodeNumDim;
    int gThreadsPerBlock = 256;
    int gBlocksPerGrid = (gNum + gThreadsPerBlock - 1) / gThreadsPerBlock;
    VelUpdate<<<gBlocksPerGrid, gThreadsPerBlock>>>(gNum, dt, ext_gravity,
                                                    mGrid.originCorner[0] + 10 * mGrid.h,
                                                    mGrid.originCorner[1] + 10 * mGrid.h,
                                                    mGrid.originCorner[2] + 10 * mGrid.h,
                                                    mGrid.originCorner[0] + mGrid.h * mGrid.nodeNumDim - 10 * mGrid.h,
                                                    mGrid.originCorner[1] + mGrid.h * mGrid.nodeNumDim - 10 * mGrid.h,
                                                    mGrid.originCorner[2] + mGrid.h * mGrid.nodeNumDim - 10 * mGrid.h,
                                                    mGrid.originCorner[0], mGrid.originCorner[1], mGrid.originCorner[2],
                                                    mGrid.nodeNumDim, mGrid.h,
                                                    mGrid.nodeMassVec, mGrid.nodeVelVec, mGrid.nodeForceVec);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch VelUpdate kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

#ifdef DEBUG

#endif

    // 5.5 Clean the velocity on the particles.
    // 6. Interpolate new velocity back to particles.
    // 7. Move particles.
    err = hipMemset(mParticles.pVelVecGRAM, 0, mParticles.velVecByteSize);
    if (err != hipSuccess){
        std::cerr << "Clean grid velocity error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    std::fill(mParticles.particleVelVec.begin(), mParticles.particleVelVec.end(), 0.0);
    InterpolateAndMove<<<pBlocksPerGrid, pThreadsPerBlock>>>(mParticles.particleNum,
                                                             dt,
                                                             mParticles.pPosVecGRAM,
                                                             mParticles.pVelVecGRAM,
                                                             mParticles.pDeformationGradientGRAM,
                                                             mGrid.originCorner[0],
                                                             mGrid.originCorner[1],
                                                             mGrid.originCorner[2],
                                                             mGrid.nodeNumDim,
                                                             mGrid.h,
                                                             mGrid.nodeVelVec);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch InterpolateAndMove kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Put particles' position and velocity back to RAM.
    err = hipMemcpy(mParticles.particlePosVec.data(), mParticles.pPosVecGRAM, mParticles.posVecByteSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy particle position memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    err = hipMemcpy(mParticles.particleVelVec.data(), mParticles.pVelVecGRAM, mParticles.velVecByteSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy particle velocity memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    t += dt;

#ifdef DEBUG
    // Check whether the energy is consistent.
    std::cout << "n+1 vel:[" << mParticles.particleVelVec[1 * 3] << " " << mParticles.particleVelVec[1 * 3 + 1] << " " << mParticles.particleVelVec[1 * 3 + 2] << "]" << std::endl;
    std::cout << "n+1 pos:[" << mParticles.particlePosVec[1 * 3] << " " << mParticles.particlePosVec[1 * 3 + 1] << " " << mParticles.particlePosVec[1 * 3 + 2] << "]" << std::endl << std::endl;
/*
    double vel_energy_cur = 0.5 * 1.0 * (mParticles.particleVelVec[1] * mParticles.particleVelVec[1]);
    double vel_energy_diff = vel_energy_cur - vel_energy_init;
    double pos0_y_cur = mParticles.particlePosVec[1];
    double gravity_energy = -1.0 * 9.8 * (pos0_y_cur - pos0_y_init);
    std::cout << "vel energy difference:" << vel_energy_diff << " gravity energy difference:" << gravity_energy << std::endl;
*/
    // Check the first particle with velocity problem:
    /*
    for (int i = 0; i < mParticles.particleNum; ++i){
        double vel_x = mParticles.particleVelVec[i * 3];
        double vel_y = mParticles.particleVelVec[i * 3 + 1];
        double vel_z = mParticles.particleVelVec[i * 3 + 2];
        if (abs(vel_x) > 0.001 || abs(vel_z) > 0.001){
            std::cout << "Problem particle id:" << i << std::endl; // 54870
            std::cout << "Problem velocity:[" << vel_x << ", " << vel_y << ", " << vel_z << "]" << std::endl;
        }
    }
    */

    /*
    FindOutBoundParticles<<<pBlocksPerGrid, pThreadsPerBlock>>>(mParticles.particleNum,
                                                                mParticles.pPosVecGRAM,
                                                                min_bound_x, min_bound_y, min_bound_z,
                                                                max_bound_x, max_bound_y, max_bound_z,
                                                                d_pAttentionLabel);
    int* h_pAttentionLabel = (int*)malloc(sizeof(int) * mParticles.particleNum);
    std::vector<int> h_pAttentionIdx;
    err = hipMemcpy(h_pAttentionLabel, d_pAttentionLabel, sizeof(int) * mParticles.particleNum, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy pAttentionLabel memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }*/

    /* Out of bound Particles -- 109743
    for (int i = 0; i < mParticles.particleNum; ++i){
        if (h_pAttentionLabel[i] == 1){
            std::cout << "Out of bound particle_idx = " << i << std::endl;
            std::cout << "f(p_idx = " << i << ") = " << "[" << h_pForce[i * 3] << ", " << h_pForce[i * 3 + 1] << ", " << h_pForce[i * 3 + 2] << "]" << std::endl;
            std::cout << "v(n+1)(p_idx = " << i << ") = [" << mParticles.particleVelVec[i * 3] << " " << mParticles.particleVelVec[i * 3 + 1] << " " << mParticles.particleVelVec[i * 3 + 2] << "]" << std::endl;
            h_pAttentionIdx.push_back(i);
        }
    }
    */

    std::cout << "***********************************" << std::endl << std::endl;
#endif
    hipFree(pForceVec);
    hipFree(gAttentionIdx);
    ++current_frame;
    current_time += dt;
}


