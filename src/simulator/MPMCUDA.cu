#include "hip/hip_runtime.h"
//
// Created by jiaruiyan on 5/28/21.
//
// Put Particle1 deformation gradient into python to check whether the stress1/2 has a problem.

#include "MPMSimulator.cuh"
#include <math.h>
#include <assert.h>
#include <thrust/device_vector.h>
#include "../../thirdparties/cudaSVD/svd3_cuda.h"


template<class T>
__device__ void Mat3x3Cofactor(const T* F, T* res){
    res[0] = F[4] * F[8] - F[5] * F[7];
    res[1] = F[5] * F[6] - F[3] * F[8];
    res[2] = F[3] * F[7] - F[4] * F[6];
    res[3] = F[2] * F[7] - F[1] * F[8];
    res[4] = F[0] * F[8] - F[2] * F[6];
    res[5] = F[1] * F[6] - F[0] * F[7];
    res[6] = F[1] * F[5] - F[2] * F[4];
    res[7] = F[2] * F[3] - F[0] * F[5];
    res[8] = F[0] * F[4] - F[1] * F[3];
}

template<class T>
__forceinline__
__device__ T Mat3x3Determinant(const T* X){
    return X[0] * (X[4] * X[8] - X[5] * X[7]) - X[1] * (X[3] * X[8] - X[5] * X[6]) + X[2] * (X[3] * X[7] - X[4] * X[6]);
}

template<class T>
__forceinline__
__device__ T Mat3x3Inv(const T* X, T* X_inv){
    T det_X = Mat3x3Determinant(X);
    if (det_X == 0.f){
        printf("Matrix inverse error.");
    }else{
        T inv_det_X = 1.f / det_X;
        X_inv[0] = inv_det_X * (X[4] * X[8] - X[5] * X[7]);
        X_inv[1] = inv_det_X * (X[2] * X[7] - X[1] * X[8]);
        X_inv[2] = inv_det_X * (X[1] * X[5] - X[2] * X[4]);
        X_inv[3] = inv_det_X * (X[5] * X[6] - X[3] * X[8]);
        X_inv[4] = inv_det_X * (X[0] * X[8] - X[2] * X[6]);
        X_inv[5] = inv_det_X * (X[2] * X[3] - X[0] * X[5]);
        X_inv[6] = inv_det_X * (X[3] * X[7] - X[4] * X[6]);
        X_inv[7] = inv_det_X * (X[1] * X[6] - X[0] * X[7]);
        X_inv[8] = inv_det_X * (X[0] * X[4] - X[1] * X[3]);
    }
}

template<class T>
__forceinline__
__device__ void MatMul3x3(const T* A, const T* B, T* C){
    C[0] = A[0] * B[0] + A[1] * B[3] + A[2] * B[6];
    C[1] = A[0] * B[1] + A[1] * B[4] + A[2] * B[7];
    C[2] = A[0] * B[2] + A[1] * B[5] + A[2] * B[8];
    C[3] = A[3] * B[0] + A[4] * B[3] + A[5] * B[6];
    C[4] = A[3] * B[1] + A[4] * B[4] + A[5] * B[7];
    C[5] = A[3] * B[2] + A[4] * B[5] + A[5] * B[8];
    C[6] = A[6] * B[0] + A[7] * B[3] + A[8] * B[6];
    C[7] = A[6] * B[1] + A[7] * B[4] + A[8] * B[7];
    C[8] = A[6] * B[2] + A[7] * B[5] + A[8] * B[8];
}

template<class T>
__forceinline__
__device__ void MatTranspose(const T* x, T* transpose) {
    transpose[0]=x[0]; transpose[1]=x[3]; transpose[2]=x[6];
    transpose[3]=x[1]; transpose[4]=x[4]; transpose[5]=x[7];
    transpose[6]=x[2]; transpose[7]=x[5]; transpose[8]=x[8];
}

__device__ void FixedCorotatedPStressSigma(float sigma1, float sigma2, float sigma3,
                                           float mu, float lambda,
                                           float& dig1, float& dig2, float& dig3){
    dig1 = 2.f * mu * (sigma1 - 1.f) + lambda * (sigma1 * sigma2 * sigma3 - 1.f) * sigma2 * sigma3;
    dig2 = 2.f * mu * (sigma2 - 1.f) + lambda * (sigma1 * sigma2 * sigma3 - 1.f) * sigma1 * sigma3;
    dig3 = 2.f * mu * (sigma3 - 1.f) + lambda * (sigma1 * sigma2 * sigma3 - 1.f) * sigma1 * sigma2;
}

__device__ void PolarSVD(float F11, float F12, float F13,
                         float F21, float F22, float F23,
                         float F31, float F32, float F33,
                         float &U11, float &U12, float &U13,
                         float &U21, float &U22, float &U23,
                         float &U31, float &U32, float &U33,	// output U
                         float &S11, float &S22, float &S33,	// output S
                         float &V11, float &V12, float &V13,
                         float &V21, float &V22, float &V23,
                         float &V31, float &V32, float &V33	// output V
                         ){

    svd(F11, F12, F13, F21, F22, F23, F31, F32, F33,
        U11, U12, U13, U21, U22, U23, U31, U32, U33,
        S11, S22, S33,
        V11, V12, V13, V21, V22, V23, V31, V32, V33);

    float V[9] = {V11, V12, V13,
                  V21, V22, V23,
                  V31, V32, V33};
    float U[9] = {U11, U12, U13,
                  U21, U22, U23,
                  U31, U32, U33};

    if (Mat3x3Determinant(U) < 0.f){
        U[2] = -U[2];
        U[5] = -U[5];
        U[8] = -U[8];
        S33 = -S33;
        assert(Mat3x3Determinant(U) >= 0.f);
    }

    if (Mat3x3Determinant(V) < 0.f){
        V[2] = -V[2];
        V[5] = -V[5];
        V[8] = -V[8];
        S33 = -S33;
        assert(Mat3x3Determinant(V) >= 0.f);
    }

    V11 = V[0]; V12 = V[1]; V13 = V[2];
    V21 = V[3]; V22 = V[4]; V23 = V[5];
    V31 = V[6]; V32 = V[7]; V33 = V[8];

    U11 = U[0]; U12 = U[1]; U13 = U[2];
    U21 = U[3]; U22 = U[4]; U23 = U[5];
    U31 = U[6]; U32 = U[7]; U33 = U[8];
}


__device__ void FixedCorotatedPStress(float F11, float F12, float F13,
                                      float F21, float F22, float F23,
                                      float F31, float F32, float F33,
                                      float mu, float lambda,
                                      float &P11, float &P12, float &P13,
                                      float &P21, float &P22, float &P23,
                                      float &P31, float &P32, float &P33){
    float U11, U12, U13, U21, U22, U23, U31, U32, U33;
    float V11, V12, V13, V21, V22, V23, V31, V32, V33;
    float S11, S22, S33;
    float dig1, dig2, dig3;

    PolarSVD(F11, F12, F13, F21, F22, F23, F31, F32, F33,
        U11, U12, U13, U21, U22, U23, U31, U32, U33,
        S11, S22, S33,
        V11, V12, V13, V21, V22, V23, V31, V32, V33);

    float V[9] = {V11, V12, V13,
                  V21, V22, V23,
                  V31, V32, V33};
    float U[9] = {U11, U12, U13,
                  U21, U22, U23,
                  U31, U32, U33};

    FixedCorotatedPStressSigma(S11, S22, S33, mu, lambda, dig1, dig2, dig3);

    float P_sigma[9] = {dig1, 0.f, 0.f,
                        0.f, dig2, 0.f,
                        0.f, 0.f, dig3};
    float V_transpose[9];
    float tmpMat[9] = {0.f};
    float res[9] = {0.f};
    MatTranspose(V, V_transpose);
    MatMul3x3(P_sigma, V_transpose, tmpMat);
    MatMul3x3(U, tmpMat, res);
    P11 = res[0];
    P12 = res[1];
    P13 = res[2];
    P21 = res[3];
    P22 = res[4];
    P23 = res[5];
    P31 = res[6];
    P32 = res[7];
    P33 = res[8];
}

template<class T>
__device__ void MatAdd(const T* m1, const T* m2, T* mAdd, int eleNum){
    for (int i = 0; i < eleNum; ++i){
        mAdd[i] = m1[i] + m2[i];
    }
}

template<class T>
__device__ void MatMin(const T* m1, const T* m2, T* mMin, int eleNum){
    for (int i = 0; i < eleNum; ++i){
        mMin[i] = m1[i] - m2[i];
    }
}

template<class T>
__forceinline__
__device__ __host__ void MatVelMul3x3x3x1(const T* X, const T* V, T* R)
{
    R[0] = X[0] * V[0] + X[1] * V[1] + X[2] * V[2];
    R[1] = X[3] * V[0] + X[4] * V[1] + X[5] * V[2];
    R[2] = X[6] * V[0] + X[7] * V[1] + X[8] * V[2];
}


template<class T>
__device__ void ScalarMatMul(const T scalar, const T* mat, T* res, int matEleNum){
    for (int i = 0; i < matEleNum; ++i){
        res[i] = scalar * mat[i];
    }
}

template<class T>
__forceinline__
__device__ void OuterProduct(const T* v1, const T* v2, T* res){
    res[0] = v1[0] * v2[0];
    res[1] = v1[0] * v2[1];
    res[2] = v1[0] * v2[2];
    res[3] = v1[1] * v2[0];
    res[4] = v1[1] * v2[1];
    res[5] = v1[1] * v2[2];
    res[6] = v1[2] * v2[0];
    res[7] = v1[2] * v2[1];
    res[8] = v1[2] * v2[2];
}


__device__ void kirchoff_FCR(const float* F, const float mu, const float lam, float* tau){
    float U11, U12, U13, U21, U22, U23, U31, U32, U33;
    float V11, V12, V13, V21, V22, V23, V31, V32, V33;
    float S11, S22, S33;
    PolarSVD(F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
             U11, U12, U13, U21, U22, U23, U31, U32, U33,
             S11, S22, S33,
             V11, V12, V13, V21, V22, V23, V31, V32, V33);
    float V[9] = {V11, V12, V13,
                  V21, V22, V23,
                  V31, V32, V33};
    float U[9] = {U11, U12, U13,
                  U21, U22, U23,
                  U31, U32, U33};
    float V_transpose[9] = {0.f};
    MatTranspose(V, V_transpose);
    float R[9] = {0.f};
    MatMul3x3(U, V_transpose, R);
    float F_min_R[9] = {0.f};
    MatMin(F, R, F_min_R, 9);
    float F_transpose[9] = {0.f};
    MatTranspose(F, F_transpose);
    float F_min_R_FT[9] = {0.f};
    MatMul3x3(F_min_R, F_transpose, F_min_R_FT);
    float tau_term1[9] = {0.f};
    ScalarMatMul(2.f * mu, F_min_R_FT, tau_term1, 9);
    float J = Mat3x3Determinant(F);
    float tau_term2[9] = {0.f};
    float I[9] = {
            1.f, 0.f, 0.f,
            0.f, 1.f, 0.f,
            0.f, 0.f, 1.f
    };
    ScalarMatMul(lam * J * (J - 1.f), I, tau_term2, 9);
    MatAdd(tau_term1, tau_term2, tau, 9);
}


__device__ double BSplineInterpolation1DDerivative(const double x){
    if (x > -0.5 && x < 0.5){
        return -2.0 * x;
    }else if (x >= 0.5 && x < 1.5){
        return x - 1.5;
    }else if (x > -1.5 && x <= -0.5){
        return 1.5 + x;
    }else{
        return 0.0;
    }
}

__device__ double BSplineInterpolation1D(const double x){
    double abs_x = abs(x);
    if (abs_x >= 0 && abs_x < 0.5){
        return 0.75 - abs_x * abs_x;
    }
    else if (abs_x >= 0.5 && abs_x < 1.5){
        return 0.5 * (1.5 - abs_x) * (1.5 - abs_x);
    }
    else{
        return 0.0;
    }
}

__device__ void BSplineInterpolationGradient(const double xp[3], const double xi[3], const double h,
                                             double& gx, double& gy, double&gz){
    double h_inv = 1.0 / h;
    double i1 = h_inv * (xp[0] - xi[0]);
    double i2 = h_inv * (xp[1] - xi[1]);
    double i3 = h_inv * (xp[2] - xi[2]);
    gx = h_inv * BSplineInterpolation1DDerivative(i1) * BSplineInterpolation1D(i2) * BSplineInterpolation1D(i3);
    gy = h_inv * BSplineInterpolation1D(i1) * BSplineInterpolation1DDerivative(i2) * BSplineInterpolation1D(i3);
    gz = h_inv * BSplineInterpolation1D(i1) * BSplineInterpolation1D(i2) * BSplineInterpolation1DDerivative(i3);
}

__device__ double BSplineInterpolation(const double xp[3], const double xi[3], const double h){
    // printf("Interpolation:(%f, %f, %f)\n", (xp[0] - xi[0]) / h, (xp[1] - xi[1]) / h, (xp[2] - xi[2]) / h);
    return BSplineInterpolation1D((xp[0] - xi[0]) / h) *
           BSplineInterpolation1D((xp[1] - xi[1]) / h) *
           BSplineInterpolation1D((xp[2] - xi[2]) / h);
}

__global__ void P2G(unsigned int pNum, double pMass, double pVol, int pType,
                    double* pPosVec, double* pVelVec,
                    double* pEDGVec, double* pPDGVec,
                    double* pAffineVelVec,
                    double gOriCorner_x, double gOriCorner_y, double gOriCorner_z, // int* gAttentionIdx,
                    unsigned int gNodeNumDim, double h, double dt, double mu, double lambda,
                    double* gNodeMassVec, double* gNodeTmpMotVec){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < pNum){
        double pos[3] = {pPosVec[i * 3], pPosVec[i * 3 + 1], pPosVec[i * 3 + 2]};
        double m = pMass;
        double vel[3] = {pVelVec[i * 3], pVelVec[i * 3 + 1], pVelVec[i * 3 + 2]};
        double tmpAffineVel[9] = {pAffineVelVec[9 * i], pAffineVelVec[9 * i + 1], pAffineVelVec[9 * i + 2],
                                  pAffineVelVec[9 * i + 3], pAffineVelVec[9 * i + 4], pAffineVelVec[9 * i + 5],
                                  pAffineVelVec[9 * i + 6], pAffineVelVec[9 * i + 7], pAffineVelVec[9 * i + 8]};
        float tau[9] = {0.f};
        if (pType == JELLO){
            float tmpDeformationGradient[9] = {float(pEDGVec[9 * i]), float(pEDGVec[9 * i + 1]), float(pEDGVec[9 * i + 2]),
                                               float(pEDGVec[9 * i + 3]), float(pEDGVec[9 * i + 4]), float(pEDGVec[9 * i + 5]),
                                               float(pEDGVec[9 * i + 6]), float(pEDGVec[9 * i + 7]), float(pEDGVec[9 * i + 8])};
            kirchoff_FCR(tmpDeformationGradient, float(mu), float(lambda), tau);
        }else if (pType == SNOW){
            float tmpEDG[9] = {float(pEDGVec[9 * i]), float(pEDGVec[9 * i + 1]), float(pEDGVec[9 * i + 2]),
                               float(pEDGVec[9 * i + 3]), float(pEDGVec[9 * i + 4]), float(pEDGVec[9 * i + 5]),
                               float(pEDGVec[9 * i + 6]), float(pEDGVec[9 * i + 7]), float(pEDGVec[9 * i + 8])};
            float tmpPDG[9] = {float(pPDGVec[9 * i]), float(pPDGVec[9 * i + 1]), float(pPDGVec[9 * i + 2]),
                               float(pPDGVec[9 * i + 3]), float(pPDGVec[9 * i + 4]), float(pPDGVec[9 * i + 5]),
                               float(pPDGVec[9 * i + 6]), float(pPDGVec[9 * i + 7]), float(pPDGVec[9 * i + 8])};

            float Jp = Mat3x3Determinant(tmpPDG);
            float Je = Mat3x3Determinant(tmpEDG);
            float xi = 10.f;
            float harding_coef = expf(xi * (1.f - Jp));
            float snow_mu = float(mu) * harding_coef;
            float snow_lambda = float(lambda) * harding_coef;

            /*
            FixedCorotatedPStress(tmpEDG[0], tmpEDG[1], tmpEDG[2],
                                  tmpEDG[3], tmpEDG[4], tmpEDG[5],
                                  tmpEDG[6], tmpEDG[7], tmpEDG[8],
                                  snow_mu, snow_lambda,
                                  stress[0], stress[1], stress[2],
                                  stress[3], stress[4], stress[5],
                                  stress[6], stress[7], stress[8]);
            */

            kirchoff_FCR(tmpEDG, snow_mu, snow_lambda, tau);

            /*
            if (i == 1){
                printf("Particle 1 Jp:%f, Je:%f, hard coef:%f \n, stress:\n[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n\n",
                       Jp, Je, harding_coef,
                       stress[0], stress[1], stress[2],
                       stress[3], stress[4], stress[5],
                       stress[6], stress[7], stress[8]);
            }

            if (harding_coef < 1.f){
                printf("Particle %d's hard coef:%f\n", i, harding_coef);
            }
            */
        }


        /*
        if (i == 1){
            printf("P2G: Tmp Mat of particle 1:\n");
            printf("[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n",
                   tmpMat[0], tmpMat[1], tmpMat[2],
                   tmpMat[3], tmpMat[4], tmpMat[5],
                   tmpMat[6], tmpMat[7], tmpMat[8]);
            printf("P2G: Stress of particle 1:\n");
            printf("[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n",
                   stress[0], stress[1], stress[2],
                   stress[3], stress[4], stress[5],
                   stress[6], stress[7], stress[8]);
            printf("P2G: Stress2 of particle 1:\n");
            printf("[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n",
                   stress2[0], stress2[1], stress2[2],
                   stress2[3], stress2[4], stress2[5],
                   stress2[6], stress2[7], stress2[8]);
            printf("P2G: F1:\n");
            printf("[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n",
                   tmpDeformationGradient[0], tmpDeformationGradient[1], tmpDeformationGradient[2],
                   tmpDeformationGradient[3], tmpDeformationGradient[4], tmpDeformationGradient[5],
                   tmpDeformationGradient[6], tmpDeformationGradient[7], tmpDeformationGradient[8]);
            printf("P2G: F1 determinant:%f\n", Mat3x3Determinant(tmpDeformationGradient));
            printf("mu:%f, lambda:%f\n", mu, lambda);
        }
        */

        int b_idx_x = max(0, int((pos[0] - gOriCorner_x - 0.5 * h) / h));
        int b_idx_y = max(0, int((pos[1] - gOriCorner_y - 0.5 * h) / h));
        int b_idx_z = max(0, int((pos[2] - gOriCorner_z - 0.5 * h) / h));
        double t_w = 0.0;
        double t_m = 0.0;
        float t_f_x = 0.0;
        float t_f_y = 0.0;
        float t_f_z = 0.0;
        for (int idx_x_offset = 0; idx_x_offset < 3; ++idx_x_offset){
            for (int idx_y_offset = 0; idx_y_offset < 3; ++idx_y_offset){
                for (int idx_z_offset = 0; idx_z_offset < 3; ++idx_z_offset){
                    int idx_x = b_idx_x + idx_x_offset;
                    int idx_y = b_idx_y + idx_y_offset;
                    int idx_z = b_idx_z + idx_z_offset;
                    double b_pos[3] = {gOriCorner_x + idx_x * h,
                                       gOriCorner_y + idx_y * h,
                                       gOriCorner_z + idx_z * h};
                    // printf("b_pos:(%f, %f, %f)\n", b_pos[0], b_pos[1], b_pos[2]);
                    double w = BSplineInterpolation(pos, b_pos, h);
                    int g_idx = idx_z * gNodeNumDim * gNodeNumDim + idx_y * gNodeNumDim + idx_x;
                    if (idx_x >= gNodeNumDim || idx_y >= gNodeNumDim || idx_z >= gNodeNumDim){
                        printf("Particle ID:%d, idx_z:%d, idx_y:%d, idx_x:%d\n", i, idx_z, idx_y, idx_x);
                        assert(idx_x < gNodeNumDim);
                        assert(idx_y < gNodeNumDim);
                        assert(idx_z < gNodeNumDim);
                    }
                    if (g_idx >= gNodeNumDim * gNodeNumDim * gNodeNumDim){
                        printf("Particle ID:%d, idx_z:%d, idx_y:%d, idx_x:%d", i, idx_z, idx_y, idx_x);
                    }
                    assert(g_idx < gNodeNumDim * gNodeNumDim * gNodeNumDim);
                    assert(g_idx >= 0);

                    atomicAdd(&gNodeMassVec[g_idx], w * m);
                    t_m += w * m;

                    double xi_min_xp[3] = {b_pos[0] - pos[0],
                                           b_pos[1] - pos[1],
                                           b_pos[2] - pos[2]};
                    double Cp_xi_xp[3] = {0.0};
                    MatVelMul3x3x3x1(tmpAffineVel, xi_min_xp, Cp_xi_xp);
                    double right_term[3] = {vel[0] + Cp_xi_xp[0],
                                            vel[1] + Cp_xi_xp[1],
                                            vel[2] + Cp_xi_xp[2]};
                    atomicAdd(&gNodeTmpMotVec[3 * g_idx], w * m * right_term[0]);
                    atomicAdd(&gNodeTmpMotVec[3 * g_idx + 1], w * m * right_term[1]);
                    atomicAdd(&gNodeTmpMotVec[3 * g_idx + 2], w * m * right_term[2]);
                    t_w += w;

                    /*
                    if (i == 1){
                        // printf("P109743 related node Mot(g_idx=%d)=[%f, %f, %f] w=%f.\n", g_idx,
                        //        w * m * vel[0], w * m * vel[1], w * m * vel[2], w);
                        gAttentionIdx[idx_x_offset * 9 + idx_y_offset * 3 + idx_z_offset] = g_idx;
                    }
                    */

                    // Transfer elasticity force to grid.
                    double grad_wip[3] = {0.0};
                    float tmpForce[3] = {0.f};
                    BSplineInterpolationGradient(pos, b_pos, h, grad_wip[0], grad_wip[1], grad_wip[2]);
                    float grad_wip_f[3] = {static_cast<float>(grad_wip[0]), static_cast<float>(grad_wip[1]), static_cast<float>(grad_wip[2])};
                    MatVelMul3x3x3x1(tau, grad_wip_f, tmpForce);
                    ScalarMatMul(-float(pVol), tmpForce, tmpForce, 3);
                    atomicAdd(&gNodeTmpMotVec[3 * g_idx], dt * tmpForce[0]);
                    atomicAdd(&gNodeTmpMotVec[3 * g_idx + 1], dt * tmpForce[1]);
                    atomicAdd(&gNodeTmpMotVec[3 * g_idx + 2], dt * tmpForce[2]);
                    /*
                    if (i == 1){
                        printf("Particle %d contributes force to g_idx = %d:[%f, %f, %f]\n", i, g_idx, -tmpForce[0], -tmpForce[1], -tmpForce[2]);
                        printf("Particle %d contributes grad_wip = %d:[%f, %f, %f]\n", i, g_idx, grad_wip_f[0], grad_wip_f[1], grad_wip_f[2]);
                        // pAttentionLabel[i] = 1;
                    }
                    */

                    t_f_x -= tmpForce[0];
                    t_f_y -= tmpForce[1];
                    t_f_z -= tmpForce[2];

                    /*
                    if (i == 0){
                        printf("grad_wip_f:(%f, %f, %f)\n", grad_wip_f[0], grad_wip_f[1], grad_wip_f[2]);
                        printf("tmpForce:(%f, %f, %f)\n", tmpForce[0], tmpForce[1], tmpForce[2]);
                        printf("total force:(%f, %f, %f)\n", t_f_x, t_f_y, t_f_z);
                    }
                    */
                }
            }
        }
        // pForceVec[3 * i] = double(t_f_x);
        // pForceVec[3 * i + 1] = double(t_f_y);
        // pForceVec[3 * i + 2] = double(t_f_z);
        /*
        if (abs(t_f_x) > 1.0 || abs(t_f_y) > 1.0 || abs(t_f_z) > 1.0){
            printf("Particle %d contributes force:[%f, %f, %f]\n", i, t_f_x, t_f_y, t_f_z);
            // pAttentionLabel[i] = 1;
        }

        if (i == 1){
            printf("P1 Elasticity Force Contribution:(%f, %f, %f)\n", t_f_x, t_f_y, t_f_z);
        }
        */

        assert(abs(t_w - 1.0) < 0.001);
        // assert(abs(t_m - 1.0) < 0.001);
    }
}

__global__ void VelUpdate(unsigned int gNum, double dt, double ext_gravity,
                          double lower_x, double lower_y, double lower_z,
                          double upper_x, double upper_y, double upper_z,
                          double gOriCorner_x, double gOriCorner_y, double gOriCorner_z,
                          unsigned int gNodeNumDim, double h,
                          double* gMassVec, double* gVelMotVec){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < gNum){
        double mass = gMassVec[i];
        if (abs(mass) < DBL_EPSILON){
            gMassVec[i] = 0.0;
            gVelMotVec[3 * i] = 0.0;
            gVelMotVec[3 * i + 1] = 0.0;
            gVelMotVec[3 * i + 2] = 0.0;
        }
        else{
            // Calculate velocity from momentum.
            gVelMotVec[3 * i] = gVelMotVec[3 * i] / mass;
            gVelMotVec[3 * i + 1] = gVelMotVec[3 * i + 1] / mass;
            gVelMotVec[3 * i + 2] = gVelMotVec[3 * i + 2] / mass;

            // Include gravity into velocity.
            gVelMotVec[3 * i + 1] = gVelMotVec[3 * i + 1] + ext_gravity * dt;

            // Deal with Boundary condition.
            int idx_x = i % int(gNodeNumDim);
            int idx_y = ((i - idx_x) / int(gNodeNumDim)) % int(gNodeNumDim);
            int idx_z = ((i - idx_x) / int(gNodeNumDim) - idx_y) / int(gNodeNumDim);
            double grid_node_pos[3] = {gOriCorner_x + idx_x * h,
                                       gOriCorner_y + idx_y * h,
                                       gOriCorner_z + idx_z * h};
            if (grid_node_pos[0] <= lower_x || grid_node_pos[0] >= upper_x || grid_node_pos[1] <= lower_y ||
                grid_node_pos[1] >= upper_y || grid_node_pos[2] <= lower_z || grid_node_pos[2] >= upper_z){
                gVelMotVec[3 * i] = 0.0;
                gVelMotVec[3 * i + 1] = 0.0;
                gVelMotVec[3 * i + 2] = 0.0;
            }
        }
    }
}

__global__ void InterpolateAndMove(unsigned int pNum, double dt, int pType,
                                   double* pPosVec, double* pVelVec,
                                   double* pEDGVec, double* pPDGVec,
                                   double* pAffineVelVec,
                                   double* pDGDiffVec,
                                   double gOriCorner_x, double gOriCorner_y, double gOriCorner_z,
                                   unsigned int gNodeNumDim, double h, double* gNodeVelVec){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < pNum){
        double h2_inv = 1.0 / (h * h);
        double pos[3] = {pPosVec[i * 3], pPosVec[i * 3 + 1], pPosVec[i * 3 + 2]};
        int b_idx_x = max(0, int((pos[0] - gOriCorner_x - 0.5 * h) / h));
        int b_idx_y = max(0, int((pos[1] - gOriCorner_y - 0.5 * h) / h));
        int b_idx_z = max(0, int((pos[2] - gOriCorner_z - 0.5 * h) / h));
        double t_w = 0.0;
        double t_vel_x = 0.0;
        double t_vel_y = 0.0;
        double t_vel_z = 0.0;
        double res_affine_vel[9] = {
                pAffineVelVec[9 * i], pAffineVelVec[9 * i + 1], pAffineVelVec[9 * i + 2],
                pAffineVelVec[9 * i + 3], pAffineVelVec[9 * i + 4], pAffineVelVec[9 * i + 5],
                pAffineVelVec[9 * i + 6], pAffineVelVec[9 * i + 7], pAffineVelVec[9 * i + 8],
        };

        for (int idx_x_offset = 0; idx_x_offset < 3; ++idx_x_offset){
            for (int idx_y_offset = 0; idx_y_offset < 3; ++idx_y_offset){
                for (int idx_z_offset = 0; idx_z_offset < 3; ++idx_z_offset){
                    int idx_x = b_idx_x + idx_x_offset;
                    int idx_y = b_idx_y + idx_y_offset;
                    int idx_z = b_idx_z + idx_z_offset;
                    double b_pos[3] = {gOriCorner_x + idx_x * h,
                                       gOriCorner_y + idx_y * h,
                                       gOriCorner_z + idx_z * h};

                    double w = BSplineInterpolation(pos, b_pos, h);
                    int g_idx = idx_z * gNodeNumDim * gNodeNumDim + idx_y * gNodeNumDim + idx_x;
                    assert(g_idx < gNodeNumDim * gNodeNumDim * gNodeNumDim);
                    assert(g_idx >= 0);
                    double gVel[3] = {gNodeVelVec[3 * g_idx], gNodeVelVec[3 * g_idx + 1], gNodeVelVec[3 * g_idx + 2]};

                    pVelVec[3 * i] += w * gVel[0];
                    pVelVec[3 * i + 1] += w * gVel[1];
                    pVelVec[3 * i + 2] += w * gVel[2];

                    double xi_min_xp[3] = {b_pos[0] - pos[0],
                                           b_pos[1] - pos[1],
                                           b_pos[2] - pos[2]};
                    double vi_xi_xp[9] = {0.0};
                    OuterProduct(gVel, xi_min_xp, vi_xi_xp);
                    double acc_term[9] = {0.0};
                    ScalarMatMul(4.0 * w * h2_inv, vi_xi_xp, acc_term, 9);
                    double prev_affine_vel[9] = {
                            res_affine_vel[0], res_affine_vel[1], res_affine_vel[2],
                            res_affine_vel[3], res_affine_vel[4], res_affine_vel[5],
                            res_affine_vel[6], res_affine_vel[7], res_affine_vel[8]
                    };
                    MatAdd(prev_affine_vel, acc_term, res_affine_vel, 9);

                    t_vel_x += gVel[0];
                    t_vel_y += gVel[1];
                    t_vel_z += gVel[2];
                    // printf("gNode vel:[%f, %f, %f]\n", gNodeVelVec[3 * g_idx], gNodeVelVec[3 * g_idx + 1], gNodeVelVec[3 * g_idx + 2]);
                    t_w += w;
                }
            }
        }
        assert(abs(t_w - 1.0) < 0.0001);
        double vel_p[3] = {pVelVec[3 * i], pVelVec[3 * i + 1], pVelVec[3 * i + 2]};
        pAffineVelVec[9 * i] = res_affine_vel[0];
        pAffineVelVec[9 * i + 1] = res_affine_vel[1];
        pAffineVelVec[9 * i + 2] = res_affine_vel[2];
        pAffineVelVec[9 * i + 3] = res_affine_vel[3];
        pAffineVelVec[9 * i + 4] = res_affine_vel[4];
        pAffineVelVec[9 * i + 5] = res_affine_vel[5];
        pAffineVelVec[9 * i + 6] = res_affine_vel[6];
        pAffineVelVec[9 * i + 7] = res_affine_vel[7];
        pAffineVelVec[9 * i + 8] = res_affine_vel[8];

        // Update deformation gradient
        double grad_v[9] = {0.0};
        for (int idx_x_offset = 0; idx_x_offset < 3; ++idx_x_offset){
            for (int idx_y_offset = 0; idx_y_offset < 3; ++idx_y_offset){
                for (int idx_z_offset = 0; idx_z_offset < 3; ++idx_z_offset){
                    int idx_x = b_idx_x + idx_x_offset;
                    int idx_y = b_idx_y + idx_y_offset;
                    int idx_z = b_idx_z + idx_z_offset;
                    double b_pos[3] = {gOriCorner_x + idx_x * h,
                                       gOriCorner_y + idx_y * h,
                                       gOriCorner_z + idx_z * h};
                    int g_idx = idx_z * gNodeNumDim * gNodeNumDim + idx_y * gNodeNumDim + idx_x;
                    assert(g_idx < gNodeNumDim * gNodeNumDim * gNodeNumDim);
                    assert(g_idx >= 0);

                    double grad_wip[3] = {0.0};
                    BSplineInterpolationGradient(pos, b_pos, h, grad_wip[0], grad_wip[1], grad_wip[2]);

                    double vi[3] = {gNodeVelVec[3 * g_idx], gNodeVelVec[3 * g_idx + 1], gNodeVelVec[3 * g_idx + 2]};

                    double add_mat[9] = {0.0};
                    OuterProduct(vi, grad_wip, add_mat);

                    double tmp_grad_v[9];
                    memcpy(tmp_grad_v, grad_v, sizeof(double) * 9);
                    MatAdd(tmp_grad_v, add_mat, grad_v, 9);
                }
            }
        }

        double leftMat[9] = {1.0, 0.0, 0.0,
                             0.0, 1.0, 0.0,
                             0.0, 0.0, 1.0};
        double tmp_leftMat[9] = {1.0, 0.0, 0.0,
                                 0.0, 1.0, 0.0,
                                 0.0, 0.0, 1.0};
        ScalarMatMul(dt, grad_v, grad_v, 9);
        MatAdd(tmp_leftMat, grad_v, leftMat, 9);

        if (pType == JELLO){
            double Fp[9] = {pEDGVec[9 * i], pEDGVec[9 * i + 1], pEDGVec[9 * i + 2],
                            pEDGVec[9 * i + 3], pEDGVec[9 * i + 4], pEDGVec[9 * i + 5],
                            pEDGVec[9 * i + 6], pEDGVec[9 * i + 7], pEDGVec[9 * i + 8]};

            double tmp_Fp[9];
            memcpy(tmp_Fp, Fp, sizeof(double) * 9);
            MatMul3x3(leftMat, tmp_Fp, Fp);
            pEDGVec[9 * i] = Fp[0];
            pEDGVec[9 * i + 1] = Fp[1];
            pEDGVec[9 * i + 2] = Fp[2];
            pEDGVec[9 * i + 3] = Fp[3];
            pEDGVec[9 * i + 4] = Fp[4];
            pEDGVec[9 * i + 5] = Fp[5];
            pEDGVec[9 * i + 6] = Fp[6];
            pEDGVec[9 * i + 7] = Fp[7];
            pEDGVec[9 * i + 8] = Fp[8];
            double dgDet = Mat3x3Determinant(Fp);
            assert(dgDet > 0);
            if (dgDet > 1.0){
                pDGDiffVec[i] = dgDet;
            }else{
                pDGDiffVec[i] = 1.0 / dgDet;
            }
        }else if (pType == SNOW){
            double eF_n[9] = {
                    pEDGVec[9 * i], pEDGVec[9 * i + 1], pEDGVec[9 * i + 2],
                    pEDGVec[9 * i + 3], pEDGVec[9 * i + 4], pEDGVec[9 * i + 5],
                    pEDGVec[9 * i + 6], pEDGVec[9 * i + 7], pEDGVec[9 * i + 8]
            };
            double pF_n[9] = {
                    pPDGVec[9 * i], pPDGVec[9 * i + 1], pPDGVec[9 * i + 2],
                    pPDGVec[9 * i + 3], pPDGVec[9 * i + 4], pPDGVec[9 * i + 5],
                    pPDGVec[9 * i + 6], pPDGVec[9 * i + 7], pPDGVec[9 * i + 8]
            };
            double eF_nplus1_trial[9] = {0.0};
            MatMul3x3(leftMat, eF_n, eF_nplus1_trial);

            float Uf[9] = {0.f};
            float Vf[9] = {0.f};
            float Etaf[9] = {0.f};
            double Eta_star[9] = {0.0};
            PolarSVD(float(eF_nplus1_trial[0]), float(eF_nplus1_trial[1]), float(eF_nplus1_trial[2]),
                float(eF_nplus1_trial[3]), float(eF_nplus1_trial[4]), float(eF_nplus1_trial[5]),
                float(eF_nplus1_trial[6]), float(eF_nplus1_trial[7]), float(eF_nplus1_trial[8]),
                Uf[0], Uf[1], Uf[2], Uf[3], Uf[4], Uf[5], Uf[6], Uf[7], Uf[8],
                Etaf[0], Etaf[4], Etaf[8],
                Vf[0], Vf[1], Vf[2], Vf[3], Vf[4], Vf[5], Vf[6], Vf[7], Vf[8]);

            double U[9] = {
                    double(Uf[0]), double(Uf[1]), double(Uf[2]),
                    double(Uf[3]), double(Uf[4]), double(Uf[5]),
                    double(Uf[6]), double(Uf[7]), double(Uf[8]),
            };

            double V[9] = {
                    double(Vf[0]), double(Vf[1]), double(Vf[2]),
                    double(Vf[3]), double(Vf[4]), double(Vf[5]),
                    double(Vf[6]), double(Vf[7]), double(Vf[8]),
            };

            //
            Eta_star[0] = min(max(double(Etaf[0]), 1 - 8e-2), 1 + 2e-2);
            Eta_star[4] = min(max(double(Etaf[4]), 1 - 8e-2), 1 + 2e-2);
            Eta_star[8] = min(max(double(Etaf[8]), 1 - 8e-2), 1 + 2e-2);

            double V_transpose[9] = {0.0};
            MatTranspose(V, V_transpose);

            double eF_nplus1[9] = {0.0};
            double leftTmp[9] = {0.0};
            MatMul3x3(U, Eta_star, leftTmp);
            MatMul3x3(leftTmp, V_transpose, eF_nplus1);

            double eF_nplus1_inv[9] = {0.0};
            Mat3x3Inv(eF_nplus1, eF_nplus1_inv);
            double mat_check[9] = {0.0};
            MatMul3x3(eF_nplus1, eF_nplus1_inv, mat_check);
            /*
            if (i == 1){
                printf("[%f, %f, %f]\n[%f, %f, %f]\n[%f, %f, %f]\n",
                       mat_check[0], mat_check[1], mat_check[2],
                       mat_check[3], mat_check[4], mat_check[5],
                       mat_check[6], mat_check[7], mat_check[8]);
            }
            */

            double pF_nplus1[9] = {0.0};
            double leftTmp2[9] = {0.0};
            MatMul3x3(eF_nplus1_inv, eF_nplus1_trial, leftTmp2);
            MatMul3x3(leftTmp2, pF_n, pF_nplus1);

            pEDGVec[9 * i] = eF_nplus1[0];
            pEDGVec[9 * i + 1] = eF_nplus1[1];
            pEDGVec[9 * i + 2] = eF_nplus1[2];
            pEDGVec[9 * i + 3] = eF_nplus1[3];
            pEDGVec[9 * i + 4] = eF_nplus1[4];
            pEDGVec[9 * i + 5] = eF_nplus1[5];
            pEDGVec[9 * i + 6] = eF_nplus1[6];
            pEDGVec[9 * i + 7] = eF_nplus1[7];
            pEDGVec[9 * i + 8] = eF_nplus1[8];

            pPDGVec[9 * i] = pF_nplus1[0];
            pPDGVec[9 * i + 1] = pF_nplus1[1];
            pPDGVec[9 * i + 2] = pF_nplus1[2];
            pPDGVec[9 * i + 3] = pF_nplus1[3];
            pPDGVec[9 * i + 4] = pF_nplus1[4];
            pPDGVec[9 * i + 5] = pF_nplus1[5];
            pPDGVec[9 * i + 6] = pF_nplus1[6];
            pPDGVec[9 * i + 7] = pF_nplus1[7];
            pPDGVec[9 * i + 8] = pF_nplus1[8];

            double dgDet = 0.0;
            double pdgDet = Mat3x3Determinant(pF_nplus1);
            assert(pdgDet > 0);
            if (pdgDet > 1.0){
                dgDet = max(dgDet, pdgDet);
            }else{
                dgDet = max(dgDet, 1.0 / pdgDet);
            }
            pDGDiffVec[i] = dgDet;
        }

        // Apply velocity
        pPosVec[3 * i] += dt * vel_p[0];
        pPosVec[3 * i + 1] += dt * vel_p[1];
        pPosVec[3 * i + 2] += dt * vel_p[2];

        // Check whether particle is out of the boundary
        double upperBound[3] = {gOriCorner_x + h * (gNodeNumDim - 1),
                                gOriCorner_y + h * (gNodeNumDim - 1),
                                gOriCorner_z + h * (gNodeNumDim - 1)};
        if (pPosVec[3 * i] < gOriCorner_x || pPosVec[3 * i] > upperBound[0] ||
            pPosVec[3 * i + 1] < gOriCorner_y || pPosVec[3 * i + 1] > upperBound[1] ||
            pPosVec[3 * i + 2] < gOriCorner_z || pPosVec[3 * i + 2] > upperBound[2]){
            printf("Particle ID:%d; vel=[%f, %f, %f]; pos=[%f, %f, %f]; Lowerbound=[%f, %f, %f]; UpperBound=[%f, %f, %f]\n\n", i,
                   vel_p[0], vel_p[1], vel_p[2],
                   pPosVec[3 * i], pPosVec[3 * i + 1], pPosVec[3 * i + 2],
                   gOriCorner_x, gOriCorner_y, gOriCorner_z,
                   upperBound[0], upperBound[1], upperBound[2]);
            assert(pPosVec[3 * i] >= gOriCorner_x);
            assert(pPosVec[3 * i] < upperBound[0]);
            assert(pPosVec[3 * i + 1] >= gOriCorner_y);
            assert(pPosVec[3 * i + 1] < upperBound[1]);
            assert(pPosVec[3 * i + 2] >= gOriCorner_z);
            assert(pPosVec[3 * i + 2] < upperBound[2]);
        }
    }
}

__global__ void FindAllRelatedParticles(unsigned int pNum, double* pPosVec,
                                        double gOriCorner_x, double gOriCorner_y, double gOriCorner_z,
                                        double h, unsigned int gNodeNumDim,
                                        int* gAttentionIdx,
                                        int* pAttentionParticleIdx){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < pNum){
        double pos[3] = {pPosVec[i * 3], pPosVec[i * 3 + 1], pPosVec[i * 3 + 2]};
        int b_idx_x = max(0, int((pos[0] - gOriCorner_x - 0.5 * h) / h));
        int b_idx_y = max(0, int((pos[1] - gOriCorner_y - 0.5 * h) / h));
        int b_idx_z = max(0, int((pos[2] - gOriCorner_z - 0.5 * h) / h));

        for (int idx_x_offset = 0; idx_x_offset < 3; ++idx_x_offset){
            for (int idx_y_offset = 0; idx_y_offset < 3; ++idx_y_offset){
                for (int idx_z_offset = 0; idx_z_offset < 3; ++idx_z_offset){
                    int idx_x = b_idx_x + idx_x_offset;
                    int idx_y = b_idx_y + idx_y_offset;
                    int idx_z = b_idx_z + idx_z_offset;
                    int g_idx = idx_z * gNodeNumDim * gNodeNumDim + idx_y * gNodeNumDim + idx_x;
                    for (int j = 0; j < 27; ++j){
                        if (gAttentionIdx[j] == g_idx){
                            pAttentionParticleIdx[i] = 1.0;
                            return;
                        }
                    }
                }
            }
        }
    }
}

__global__ void FindOutBoundParticles(unsigned int pNum, double* pPosVec,
                                      double minBoundX, double minBoundY, double minBoundZ,
                                      double maxBoundX, double maxBoundY, double maxBoundZ,
                                      int* pAttentionParticleLabel){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < pNum){
        double pos[3] = {pPosVec[i * 3], pPosVec[i * 3 + 1], pPosVec[i * 3 + 2]};
        if (pos[0] < minBoundX || pos[0] > maxBoundX ||
            pos[1] < minBoundY || pos[1] > maxBoundY ||
            pos[2] < minBoundZ || pos[2] > maxBoundZ){
            pAttentionParticleLabel[i] = 1;
        }
        else{
            pAttentionParticleLabel[i] = 0;
        }
    }
}

/* This can be done freely during P2G:
void FindRelatedGrid(int tarIdx, double gOriCorner_x, double gOriCorner_y, double gOriCorner_z,
                     double h, std::vector<int>& gAttentionIdx){

}
*/

void MPMSimulator::step() {

    // 0. Check each particles is affected by 9 grid nodes and within the grid.
    hipError_t err = hipSuccess;

    // 1. Clean grid data.
    err = hipMemset(mGrid.nodeMassVec, 0, mGrid.massVecByteSize);
    if(err != hipSuccess){
        std::cerr << "Clean grid mass error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    err = hipMemset(mGrid.nodeVelVec, 0, mGrid.velVecByteSize);
    if (err != hipSuccess){
        std::cerr << "Clean grid velocity error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
/*
#ifdef DEBUG
    std::cout << "********* Frame " << current_frame << " starts **********" << std::endl << std::endl;
    double standard_speed = ext_gravity * t;
    // Check particle 1 pos and vel:
    std::cout << "n vel:[" << mParticles.particleVelVec[1 * 3] << " " << mParticles.particleVelVec[1 * 3 + 1] << " " << mParticles.particleVelVec[1 * 3 + 2] << "]" << std::endl;
    std::cout << "n pos:[" << mParticles.particlePosVec[1 * 3] << " " << mParticles.particlePosVec[1 * 3 + 1] << " " << mParticles.particlePosVec[1 * 3 + 2] << "]" << std::endl << std::endl;

#endif
*/

/*
    double* pForceVec;
    err = hipMalloc(&pForceVec, mParticles.particleNum * sizeof(double) * 3);
    if (err != hipSuccess){
        std::cerr << "Allocate particle force vector error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    int* gAttentionIdx;
    err = hipMalloc(&gAttentionIdx, 27 * sizeof(int));
    if (err != hipSuccess){
        std::cerr << "Allocate attention idx error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
*/

    // 2. Transfer mass to the grid.
    // 3. Transfer velocity(Momentum) to the grid.
    for (int i = 0; i < mParticlesGroupsVec.size(); ++i){
        int pThreadsPerBlock = 128;
        int pBlocksPerGrid = (mParticlesGroupsVec[i].particleNum + pThreadsPerBlock - 1) / pThreadsPerBlock;
        P2G<<<pBlocksPerGrid, pThreadsPerBlock>>>(mParticlesGroupsVec[i].particleNum,
                                                  mParticlesGroupsVec[i].mParticleMass,
                                                  mParticlesGroupsVec[i].mParticleVolume,
                                                  mParticlesGroupsVec[i].mMaterial.mType,
                                                  mParticlesGroupsVec[i].pPosVecGRAM,
                                                  mParticlesGroupsVec[i].pVelVecGRAM,
                                                  mParticlesGroupsVec[i].pElasiticityDeformationGradientGRAM,
                                                  mParticlesGroupsVec[i].pPlasiticityDeformationGradientGRAM,
                                                  mParticlesGroupsVec[i].pAffineVelGRAM,
                                                  mGrid.originCorner[0], mGrid.originCorner[1], mGrid.originCorner[2],
                                                  mGrid.nodeNumDim,
                                                  mGrid.h, adp_dt, mParticlesGroupsVec[i].mMaterial.mMu, mParticlesGroupsVec[i].mMaterial.mLambda,
                                                  mGrid.nodeMassVec,
                                                  mGrid.nodeVelVec);
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch %d's P2G kernel (error code %s)!\n", i, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }


/*
#ifdef DEBUG
    double pos0_y_init = mParticles.particlePosVec[1];
    double vel0_y_init = mParticles.particleVelVec[1];
    double vel_energy_init = 0.5 * 1.0 * vel0_y_init * vel0_y_init;

    // Check whether grid mass is equal to particles mass.
    double* h_gMassVec = (double*)malloc(mGrid.massVecByteSize);
    err = hipMemcpy(h_gMassVec, mGrid.nodeMassVec, mGrid.massVecByteSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy grid mass memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    thrust::device_vector<double> gMass(h_gMassVec, h_gMassVec + mGrid.massVecByteSize / sizeof(double));
    thrust::device_vector<double> pMass(mParticles.particleMassVec.begin(), mParticles.particleMassVec.end());

    double gSum = thrust::reduce(gMass.begin(), gMass.end());
    double pSum = thrust::reduce(pMass.begin(), pMass.end());
    if (abs(gSum - pSum) > 0.0001){
        std::cerr << "Mass is different between Grid and particles after P2G." << std::endl;
        std::cerr << "gSum:" << gSum << " pSum:" << pSum << std::endl;
        exit(1);
    }
    free(h_gMassVec);

    // Check force on the grid.
    double* h_gForceVec = (double*)malloc(mGrid.forceVecByteSize);
    err = hipMemcpy(h_gForceVec, mGrid.nodeForceVec, mGrid.forceVecByteSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy grid force memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    int* h_gAttentionIdx = (int*)malloc(sizeof(int) * 27);
    err = hipMemcpy(h_gAttentionIdx, gAttentionIdx, sizeof(int) * 27, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy attentionIdx memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    for (int i = 0; i < 27; ++i){
        int g_idx = h_gAttentionIdx[i];
        std::cout << "f(g_idx=" << g_idx << ")" << "=[" << h_gForceVec[g_idx * 3] << ", " << h_gForceVec[g_idx * 3 + 1] << ", " << h_gForceVec[g_idx * 3 + 2] << "]" << std::endl;
    }

    int* d_pAttentionLabel;
    err = hipMalloc(&d_pAttentionLabel, mParticles.particleNum * sizeof(int));
    if (err != hipSuccess){
        std::cerr << "Allocate pAttentionLabel memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    err = hipMemset(d_pAttentionLabel, 0, mParticles.particleNum * sizeof(int));
    if (err != hipSuccess){
        std::cerr << "Clean AttentionLabel error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
*/
    /*
    FindAllRelatedParticles<<<pBlocksPerGrid, pThreadsPerBlock>>>(mParticles.particleNum,
                                                                  mParticles.pPosVecGRAM,
                                                                  mGrid.originCorner[0],
                                                                  mGrid.originCorner[1],
                                                                  mGrid.originCorner[2],
                                                                  mGrid.h,
                                                                  mGrid.nodeNumDim,
                                                                  gAttentionIdx,
                                                                  pAttentionLabel);
    */
/*
    double* h_pForce = (double*)malloc(sizeof(double) * mParticles.particleNum * 3);
    err = hipMemcpy(h_pForce, pForceVec, sizeof(double) * mParticles.particleNum * 3, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy pForceVec memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }
*/
    /* Attention Nodes:
    for (int i = 0; i < mParticles.particleNum; ++i){
        if (h_pAttentionLabel[i] == 1){
            std::cout << "Related nodes are affected by particle_idx = " << i << std::endl;
            std::cout << "f(p_idx = " << i << ") = " << "[" << h_pForce[i * 3] << ", " << h_pForce[i * 3 + 1] << ", " << h_pForce[i * 3 + 2] << "]" << std::endl;
            h_pAttentionIdx.push_back(i);
        }
    }
    */
// #endif

    // 4. Calculate the velocity.
    // 5. Apply gravity.
    int gNum = mGrid.nodeNumDim * mGrid.nodeNumDim * mGrid.nodeNumDim;
    int gThreadsPerBlock = 256;
    int gBlocksPerGrid = (gNum + gThreadsPerBlock - 1) / gThreadsPerBlock;
    VelUpdate<<<gBlocksPerGrid, gThreadsPerBlock>>>(gNum, adp_dt, ext_gravity,
                                                    mGrid.originCorner[0] + 10 * mGrid.h,
                                                    mGrid.originCorner[1] + 10 * mGrid.h,
                                                    mGrid.originCorner[2] + 10 * mGrid.h,
                                                    mGrid.originCorner[0] + mGrid.h * mGrid.nodeNumDim - 10 * mGrid.h,
                                                    mGrid.originCorner[1] + mGrid.h * mGrid.nodeNumDim - 10 * mGrid.h,
                                                    mGrid.originCorner[2] + mGrid.h * mGrid.nodeNumDim - 10 * mGrid.h,
                                                    mGrid.originCorner[0], mGrid.originCorner[1], mGrid.originCorner[2],
                                                    mGrid.nodeNumDim, mGrid.h,
                                                    mGrid.nodeMassVec, mGrid.nodeVelVec);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch VelUpdate kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

#ifdef DEBUG

#endif

    // 5.5 Clean the velocity and affine velocity on the particles.
    // 6. Interpolate new velocity back to particles.
    // 7. Move particles.
    double tmp_factor = 0.0;
    double factor = 0.0;
    for (int i = 0; i < mParticlesGroupsVec.size(); ++i){
        int pThreadsPerBlock = 128;
        int pBlocksPerGrid = (mParticlesGroupsVec[i].particleNum + pThreadsPerBlock - 1) / pThreadsPerBlock;

        err = hipMemset(mParticlesGroupsVec[i].pVelVecGRAM, 0, mParticlesGroupsVec[i].velVecByteSize);
        if (err != hipSuccess){
            std::cerr << "Clean particle velocity error." << std::endl << hipGetErrorString(err) << std::endl;
            exit(1);
        }
        err = hipMemset(mParticlesGroupsVec[i].pAffineVelGRAM, 0, mParticlesGroupsVec[i].affineVelVecByteSize);
        if (err != hipSuccess){
            std::cerr << "Clean particle affine velocity error." << std::endl << hipGetErrorString(err) << std::endl;
            exit(1);
        }

        InterpolateAndMove<<<pBlocksPerGrid, pThreadsPerBlock>>>(mParticlesGroupsVec[i].particleNum,
                                                                 adp_dt,
                                                                 mParticlesGroupsVec[i].mMaterial.mType,
                                                                 mParticlesGroupsVec[i].pPosVecGRAM,
                                                                 mParticlesGroupsVec[i].pVelVecGRAM,
                                                                 mParticlesGroupsVec[i].pElasiticityDeformationGradientGRAM,
                                                                 mParticlesGroupsVec[i].pPlasiticityDeformationGradientGRAM,
                                                                 mParticlesGroupsVec[i].pAffineVelGRAM,
                                                                 mParticlesGroupsVec[i].pDeformationGradientDiffGRAM,
                                                                 mGrid.originCorner[0],
                                                                 mGrid.originCorner[1],
                                                                 mGrid.originCorner[2],
                                                                 mGrid.nodeNumDim,
                                                                 mGrid.h,
                                                                 mGrid.nodeVelVec);
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess){
            fprintf(stderr, "Failed to launch InterpolateAndMove kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Put particles' position back to RAM.
        err = hipMemcpy(mParticlesGroupsVec[i].particlePosVec.data(),
                         mParticlesGroupsVec[i].pPosVecGRAM,
                         mParticlesGroupsVec[i].posVecByteSize,
                         hipMemcpyDeviceToHost);
        if (err != hipSuccess){
            std::cerr << "Copy particle position memory error." << std::endl << hipGetErrorString(err) << std::endl;
            exit(1);
        }

        // Time step control:
        thrust::device_ptr<double> dev_ptr_start = thrust::device_pointer_cast(mParticlesGroupsVec[i].pDeformationGradientDiffGRAM);
        thrust::device_ptr<double> dev_ptr_end = thrust::device_pointer_cast(mParticlesGroupsVec[i].pDeformationGradientDiffGRAM + mParticlesGroupsVec[i].particleNum);
        auto max_diff = thrust::max_element(dev_ptr_start, dev_ptr_end);
        tmp_factor = std::max(*max_diff * *max_diff * *max_diff, tmp_factor);
        factor = std::min(10.0, tmp_factor);
    }
    // adp_dt = max_dt / factor;

    // auto min_diff = thrust::min_element(dev_ptr_start, dev_ptr_end);
    // std::cout << "Max determinant:" << *max_diff << std::endl; //<< " Min determinant:" << *min_diff << std::endl;


#ifdef DEBUG
    // Check whether the energy is consistent.
    // std::cout << "n+1 vel:[" << mParticles.particleVelVec[1 * 3] << " " << mParticles.particleVelVec[1 * 3 + 1] << " " << mParticles.particleVelVec[1 * 3 + 2] << "]" << std::endl;
    // std::cout << "n+1 pos:[" << mParticles.particlePosVec[1 * 3] << " " << mParticles.particlePosVec[1 * 3 + 1] << " " << mParticles.particlePosVec[1 * 3 + 2] << "]" << std::endl << std::endl;
/*
    double vel_energy_cur = 0.5 * 1.0 * (mParticles.particleVelVec[1] * mParticles.particleVelVec[1]);
    double vel_energy_diff = vel_energy_cur - vel_energy_init;
    double pos0_y_cur = mParticles.particlePosVec[1];
    double gravity_energy = -1.0 * 9.8 * (pos0_y_cur - pos0_y_init);
    std::cout << "vel energy difference:" << vel_energy_diff << " gravity energy difference:" << gravity_energy << std::endl;
*/
    // Check the first particle with velocity problem:
    /*
    for (int i = 0; i < mParticles.particleNum; ++i){
        double vel_x = mParticles.particleVelVec[i * 3];
        double vel_y = mParticles.particleVelVec[i * 3 + 1];
        double vel_z = mParticles.particleVelVec[i * 3 + 2];
        if (abs(vel_x) > 0.001 || abs(vel_z) > 0.001){
            std::cout << "Problem particle id:" << i << std::endl; // 54870
            std::cout << "Problem velocity:[" << vel_x << ", " << vel_y << ", " << vel_z << "]" << std::endl;
        }
    }
    */

    /*
    FindOutBoundParticles<<<pBlocksPerGrid, pThreadsPerBlock>>>(mParticles.particleNum,
                                                                mParticles.pPosVecGRAM,
                                                                min_bound_x, min_bound_y, min_bound_z,
                                                                max_bound_x, max_bound_y, max_bound_z,
                                                                d_pAttentionLabel);
    int* h_pAttentionLabel = (int*)malloc(sizeof(int) * mParticles.particleNum);
    std::vector<int> h_pAttentionIdx;
    err = hipMemcpy(h_pAttentionLabel, d_pAttentionLabel, sizeof(int) * mParticles.particleNum, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cerr << "Copy pAttentionLabel memory error." << std::endl << hipGetErrorString(err) << std::endl;
        exit(1);
    }*/

    /* Out of bound Particles -- 109743
    for (int i = 0; i < mParticles.particleNum; ++i){
        if (h_pAttentionLabel[i] == 1){
            std::cout << "Out of bound particle_idx = " << i << std::endl;
            std::cout << "f(p_idx = " << i << ") = " << "[" << h_pForce[i * 3] << ", " << h_pForce[i * 3 + 1] << ", " << h_pForce[i * 3 + 2] << "]" << std::endl;
            std::cout << "v(n+1)(p_idx = " << i << ") = [" << mParticles.particleVelVec[i * 3] << " " << mParticles.particleVelVec[i * 3 + 1] << " " << mParticles.particleVelVec[i * 3 + 2] << "]" << std::endl;
            h_pAttentionIdx.push_back(i);
        }
    }
    */

    std::cout << "***********************************" << std::endl << std::endl;
#endif

    // hipFree(pForceVec);
    // hipFree(gAttentionIdx);
    ++current_frame;
    current_time += adp_dt;
}


